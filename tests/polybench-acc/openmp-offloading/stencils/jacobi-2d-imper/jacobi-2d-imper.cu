#include "hip/hip_runtime.h"
/**
 * jacobi2D.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <stdlib.h>
#include <stdarg.h>
#include <math.h>

#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "jacobi2D.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

// Macros to generate openmp schedule.
#include <macros.h>

// Offloading support functions.
#include <offload.h>

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define RUN_ON_CPU

DATA_TYPE *Agpu;
DATA_TYPE *Bgpu;

bool data_alloc_and_copy = false;

/* ------------------------------------------------------------- */
/* Arrays initialization. */
void init_array(int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n),
                DATA_TYPE POLYBENCH_2D(B, N, N, n, n)) {
  int i, j;

  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      A[i][j] = ((DATA_TYPE)i * (j + 2) + 10) / N;
      B[i][j] = ((DATA_TYPE)(i - 4) * (j - 1) + 11) / N;
    }
  }
}

/* ------------------------------------------------------------- */
void compareResults(int n, DATA_TYPE POLYBENCH_2D(a, N, N, n, n),
                    DATA_TYPE POLYBENCH_2D(a_outputFromGpu, N, N, n, n),
                    DATA_TYPE POLYBENCH_2D(b, N, N, n, n),
                    DATA_TYPE POLYBENCH_2D(b_outputFromGpu, N, N, n, n)) {
  int i, j, fail;
  fail = 0;

  // Compare output from CPU and GPU
  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      if (percentDiff(a[i][j], a_outputFromGpu[i][j]) >
          PERCENT_DIFF_ERROR_THRESHOLD) {
        fail++;
      }
    }
  }

  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      if (percentDiff(b[i][j], b_outputFromGpu[i][j]) >
          PERCENT_DIFF_ERROR_THRESHOLD) {
        fail++;
      }
    }
  }

  // Print results
  printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f "
         "Percent: %d\n",
         PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

/* ------------------------------------------------------------- */
/* DCE code. Must scan the entire live-out data.
         Can be used also to check the correctness of the output. */
static void print_array(int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n)) {
  int i, j;

  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      fprintf(stderr, DATA_PRINTF_MODIFIER, A[i][j]);
      if ((i * n + j) % 20 == 0)
        fprintf(stderr, "\n");
    }
  }
  fprintf(stderr, "\n");
}

/* ------------------------------------------------------------- */
void runJacobi2DCpu(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n),
                    DATA_TYPE POLYBENCH_2D(B, N, N, n, n)) {
  for (int t = 0; t < _PB_TSTEPS; t++) {
    for (int i = 1; i < _PB_N - 1; i++) {
      for (int j = 1; j < _PB_N - 1; j++) {
        B[i][j] = 0.2f * (A[i][j] + A[i][(j - 1)] + A[i][(1 + j)] +
                          A[(1 + i)][j] + A[(i - 1)][j]);
      }
    }

    for (int i = 1; i < _PB_N - 1; i++) {
      for (int j = 1; j < _PB_N - 1; j++) {
        A[i][j] = B[i][j];
      }
    }
  }
}

/* ------------------------------------------------------------- */
void jacobi2d_original(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n),
                       DATA_TYPE POLYBENCH_2D(B, N, N, n, n)) {
  /* Start timer. */
  polybench_start_instruments;

  runJacobi2DCpu(tsteps, n, A, B);

  /* Stop and print timer. */
  polybench_stop_instruments;
  // printf("CPU Time in seconds:\n");
  polybench_print_instruments;
}

/* ------------------------------------------------------------- */
/* Main computational kernel. The whole function will be timed,
         including the call and return. */
static void jacobi_2d_imper_omp_kernel(int tsteps, int n,
                                       DATA_TYPE POLYBENCH_2D(A, N, N, n, n),
                                       DATA_TYPE POLYBENCH_2D(B, N, N, n, n)) {
  int t, i, j;

	#pragma scop

	#pragma omp parallel private(i, j, t) num_threads(OPENMP_NUM_THREADS)
  {
		#pragma omp master
    {
      for (t = 0; t < _PB_TSTEPS; t++) {
				#pragma omp parallel for schedule(OPENMP_SCHEDULE_WITH_CHUNK)
        for (i = 1; i < _PB_N - 1; i++)
          for (j = 1; j < _PB_N - 1; j++)
            B[i][j] = 0.2 * (A[i][j] + A[i][j - 1] + A[i][1 + j] + A[1 + i][j] +
                             A[i - 1][j]);

				#pragma omp parallel for schedule(OPENMP_SCHEDULE_WITH_CHUNK)
        for (i = 1; i < _PB_N - 1; i++)
          for (j = 1; j < _PB_N - 1; j++)
            A[i][j] = B[i][j];
      }
    }
  }

/* 
  I put the parallel in omp parallel for directive, to fix this error:
  Errors with the old format:
         jacobi-2d-imper.c:145:17: error: work-sharing region may not be closely
   nested inside of work-sharing, critical, ordered, master or explicit task
   region
         #pragma omp for schedule(OPENMP_SCHEDULE_WITH_CHUNK)
                                                         ^
         jacobi-2d-imper.c:150:17: error: work-sharing region may not be closely
   nested inside of work-sharing, critical, ordered, master or explicit task
   region
         #pragma omp for schedule(OPENMP_SCHEDULE_WITH_CHUNK)
*/
#pragma endscop
}

/* ------------------------------------------------------------- */
void jacobi_2d_imper_omp(int tsteps, int n,
                         DATA_TYPE POLYBENCH_2D(A, N, N, n, n),
                         DATA_TYPE POLYBENCH_2D(B, N, N, n, n)) {
  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  jacobi_2d_imper_omp_kernel(tsteps, n, A, B);

  /* Stop and print timer. */
  polybench_stop_instruments;
  printf("OMP Time in seconds:\n");
  polybench_print_instruments;
}

/* ------------------------------------------------------------- */
__global__ void jacobi_cuda_kernel_1(int n, DATA_TYPE *A, DATA_TYPE *B) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if ((i >= 1) && (i < (_PB_N - 1)) && (j >= 1) && (j < (_PB_N - 1))) {
    B[i * N + j] =
        0.2f * (A[i * N + j] + A[i * N + (j - 1)] + A[i * N + (1 + j)] +
                A[(1 + i) * N + j] + A[(i - 1) * N + j]);
  }
}

/* ------------------------------------------------------------- */
__global__ void jacobi_cuda_kernel_2(int n, DATA_TYPE *A, DATA_TYPE *B) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if ((i >= 1) && (i < (_PB_N - 1)) && (j >= 1) && (j < (_PB_N - 1))) {
    A[i * N + j] = B[i * N + j];
  }
}

/* ------------------------------------------------------------- */
void jacobi2d_cuda_1(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n),
                     DATA_TYPE POLYBENCH_2D(B, N, N, n, n)) {

  if (!data_alloc_and_copy) {
    hipMalloc(&Agpu, N * N * sizeof(DATA_TYPE));
    hipMalloc(&Bgpu, N * N * sizeof(DATA_TYPE));
    hipMemcpy(Agpu, A, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(Bgpu, B, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    data_alloc_and_copy = true;
  }

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((unsigned int)ceil(((float)N) / ((float)block.x)),
            (unsigned int)ceil(((float)N) / ((float)block.y)));

  jacobi_cuda_kernel_1<<<grid, block>>>(n, Agpu, Bgpu);
  hipDeviceSynchronize();

  hipMemcpy(A, Agpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);
  hipMemcpy(B, Bgpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);
}

/* ------------------------------------------------------------- */
void jacobi2d_cuda_2(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n),
                     DATA_TYPE POLYBENCH_2D(B, N, N, n, n)) {

  if (!data_alloc_and_copy) {
    hipMalloc(&Agpu, N * N * sizeof(DATA_TYPE));
    hipMalloc(&Bgpu, N * N * sizeof(DATA_TYPE));
    hipMemcpy(Agpu, A, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(Bgpu, B, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    data_alloc_and_copy = true;
  }

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((unsigned int)ceil(((float)N) / ((float)block.x)),
            (unsigned int)ceil(((float)N) / ((float)block.y)));

  jacobi_cuda_kernel_2<<<grid, block>>>(n, Agpu, Bgpu);
  hipDeviceSynchronize();

  hipMemcpy(A, Agpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);
  hipMemcpy(B, Bgpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);
}

/* ------------------------------------------------------------- */
void copy_array(int n, DATA_TYPE POLYBENCH_2D(source, N, N, n, n),
                DATA_TYPE POLYBENCH_2D(dest, N, N, n, n)) {
  int i, j;

  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      dest[i][j] = source[i][j];
    }
  }
}

/* ------------------------------------------------------------- */
int main(int argc, char **argv) {
  /* Retrieve problem size. */
  int n = N;
  int tsteps = TSTEPS;

  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, N, n, n);
  POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, N, N, n, n);

  POLYBENCH_2D_ARRAY_DECL(A_OMP, DATA_TYPE, N, N, n, n);
  POLYBENCH_2D_ARRAY_DECL(B_OMP, DATA_TYPE, N, N, n, n);

  POLYBENCH_2D_ARRAY_DECL(A_GPU, DATA_TYPE, N, N, n, n);
  POLYBENCH_2D_ARRAY_DECL(B_GPU, DATA_TYPE, N, N, n, n);

  // void jacobi2d_cuda_1(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
  //                      DATA_TYPE POLYBENCH_2D(B,N,N,n,n))

  // Number of parameters to function.
  int n_params = 4;

  // loop 0.
  Func *ff_0 = (Func *)malloc(sizeof(Func));

  // Number of arguments + 1, the lists need to have last element NULL.
  ff_0->arg_types = (ffi_type **)malloc((n_params + 1) * sizeof(ffi_type *));
  ff_0->arg_values = (void **)malloc((n_params + 1) * sizeof(void *));

  ff_0->f = &jacobi2d_cuda_1;
  memset(&ff_0->ret_value, 0, sizeof(ff_0->ret_value));

  // return type.
  ff_0->ret_type = &ffi_type_void;

  ff_0->nargs = n_params;

  ff_0->arg_values[0] = &tsteps;
  ff_0->arg_values[1] = &n;
  ff_0->arg_values[2] = &A_GPU;
  ff_0->arg_values[3] = &A_GPU;
  ff_0->arg_values[4] = NULL;

  ff_0->arg_types[0] = &ffi_type_sint32;
  ff_0->arg_types[1] = &ffi_type_sint32;
  ff_0->arg_types[2] = &ffi_type_pointer;
  ff_0->arg_types[3] = &ffi_type_pointer;
  ff_0->arg_types[4] = NULL;

  // loop 1.
  Func *ff_1 = (Func *)malloc(sizeof(Func));

  // Number of arguments + 1, the lists need to have last element NULL.
  ff_1->arg_types = (ffi_type **)malloc((n_params + 1) * sizeof(ffi_type *));
  ff_1->arg_values = (void **)malloc((n_params + 1) * sizeof(void *));

  ff_1->f = &jacobi2d_cuda_2;
  memset(&ff_1->ret_value, 0, sizeof(ff_1->ret_value));

  // return type.
  ff_1->ret_type = &ffi_type_void;

  ff_1->nargs = n_params;

  ff_1->arg_values[0] = &tsteps;
  ff_1->arg_values[1] = &n;
  ff_1->arg_values[2] = &A_GPU;
  ff_1->arg_values[3] = &B_GPU;
  ff_1->arg_values[4] = NULL;

  ff_1->arg_types[0] = &ffi_type_sint32;
  ff_1->arg_types[1] = &ffi_type_sint32;
  ff_1->arg_types[2] = &ffi_type_pointer;
  ff_1->arg_types[3] = &ffi_type_pointer;
  ff_1->arg_types[4] = NULL;

  /*          device 0  device 1
   * loop 0   OMP       &jacobi2d_cuda_1
   * loop 1   OMP       &jacobi2d_cuda_2
   * matrix 2 x 2.
  */
  fprintf(stderr, "Creating table of target functions.\n");
  int nloops = 2;
  int ndevices = 2;

  if (create_target_functions_table(&table, nloops, ndevices)) {
    // Set up the library Functions table.
    assert(table != NULL);

    fprintf(stderr, "Declaring function in 0,1.\n");
    table[0][1][0] = *ff_0;
    fprintf(stderr, "Declaring function in 1,1.\n");
    table[1][1][0] = *ff_1;

    TablePointerFunctions = table;
    assert(TablePointerFunctions != NULL);
  }

  fprintf(stderr, "Calling init_array.\n");
  init_array(n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));

  /*Copy the original OMP.*/
  fprintf(stderr, "Copying A to A_OMP.\n");
  copy_array(n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(A_OMP));
  fprintf(stderr, "Copying B to B_OMP.\n");
  copy_array(n, POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_OMP));

  fprintf(stderr, "Copying A to A_GPU.\n");
  copy_array(n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(A_GPU));
  fprintf(stderr, "Copying B to B_GPU.\n");
  copy_array(n, POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_GPU));

  fprintf(stdout, "exp, num_threads, N, ORIG, OMP\n");

  fprintf(stdout, "OMP+OFF, %d, %d, ", OPENMP_NUM_THREADS, N);

  fprintf(stderr, "Calling Original.\n");
  jacobi2d_original(tsteps, n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));
  fprintf(stdout, ", ");

  fprintf(stderr, "Calling OMP.\n");
  jacobi_2d_imper_omp(tsteps, n, POLYBENCH_ARRAY(A_OMP),
                      POLYBENCH_ARRAY(B_OMP));
  fprintf(stdout, "\n");

  fprintf(stderr, "Calling compareResults(original, omp).\n");
  compareResults(n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(A_OMP),
                 POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_OMP));

  fprintf(stderr, "Calling compareResults(original, cuda).\n");
  compareResults(n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(A_GPU),
                 POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_GPU));

  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(A)));

  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(A_OMP);
  POLYBENCH_FREE_ARRAY(A_GPU);
  POLYBENCH_FREE_ARRAY(B);
  POLYBENCH_FREE_ARRAY(B_OMP);
  POLYBENCH_FREE_ARRAY(B_GPU);

  hipFree(Agpu);
  hipFree(Bgpu);

  return 0;
}

// polybench.c uses the OpenMP to parallelize somethings. This call were
// intercepted by hookomp.
#undef _OPENMP

#include <polybench.c>