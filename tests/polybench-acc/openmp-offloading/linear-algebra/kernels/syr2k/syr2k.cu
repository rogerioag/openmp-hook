#include "hip/hip_runtime.h"
/**
 * syr2k.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

#define POLYBENCH_TIME 1

#include "syr2k.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

#define RUN_ON_CPU

/* ------------------------------------------------------------- */
void init_arrays(int ni, int nj, DATA_TYPE *alpha, DATA_TYPE *beta,
                 DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
                 DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
                 DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni)) {
  int i, j;

  *alpha = 32412;
  *beta = 2123;

  for (i = 0; i < ni; i++) {
    for (j = 0; j < nj; j++) {
      A[i][j] = ((DATA_TYPE)i * j) / ni;
      B[i][j] = ((DATA_TYPE)i * j) / ni;
    }
  }

  for (i = 0; i < ni; i++) {
    for (j = 0; j < ni; j++) {
      C[i][j] = ((DATA_TYPE)i * j) / ni;
    }
  }
}

/* ------------------------------------------------------------- */
void copy_array(int ni, DATA_TYPE POLYBENCH_2D(C_source, NI, NI, ni, ni), DATA_TYPE POLYBENCH_2D(C_dest, NI, NI, ni, ni)) {
  int i, j;

  for (i = 0; i < ni; i++) {
    for (j = 0; j < ni; j++) {
      C_dest[i][j] = C_source[i][j];
      // printf("%4.2f - %4.2f\n", C_dest[i][j], C_source[i][j]);
    }
  }
}


/* ------------------------------------------------------------- */
void syr2kCpu(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
              DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni)) {
  int i, j, k;

  /*    C := alpha*A*B' + alpha*B*A' + beta*C */
  for (i = 0; i < _PB_NI; i++) {
    for (j = 0; j < _PB_NI; j++) {
      C[i][j] *= beta;
    }
  }

  for (i = 0; i < _PB_NI; i++) {
    for (j = 0; j < _PB_NI; j++) {
      for (k = 0; k < _PB_NJ; k++) {
        C[i][j] += alpha * A[i][k] * B[j][k];
        C[i][j] += alpha * B[i][k] * A[j][k];
      }
    }
  }
}

/* ------------------------------------------------------------- */
void syr2k_original(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
              DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni)) {
  
  /* Start timer. */
  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  syr2kCpu(ni, nj, alpha, beta, C, A, B);

  /* Stop and print timer. */
  printf("Original CPU Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;
}

/* ------------------------------------------------------------- */
/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static void syr2k_omp_kernel(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
                         DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni),
                         DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
                         DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj)) {
  int i, j, k;

  printf("NI e NJ: %d - %d\n", _PB_NI, _PB_NJ);

#pragma scop
#pragma omp parallel
  {
/*    C := alpha*A*B' + alpha*B*A' + beta*C */
#pragma omp for private(j) schedule(static)
    for (i = 0; i < _PB_NI; i++)
      for (j = 0; j < _PB_NI; j++)
        C[i][j] *= beta;
#pragma omp for private(j, k) schedule(static)
    for (i = 0; i < _PB_NI; i++)
      for (j = 0; j < _PB_NI; j++)
        for (k = 0; k < _PB_NJ; k++) {
          C[i][j] += alpha * A[i][k] * B[j][k];
          C[i][j] += alpha * B[i][k] * A[j][k];
        }
  }
#pragma endscop
}

/* ------------------------------------------------------------- */
void syr2k_omp(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
                         DATA_TYPE POLYBENCH_2D(C_outputFromOMP, NI, NI, ni, ni),
                         DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
                         DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj)) {

  /* Start timer. */
  polybench_start_instruments;

  syr2k_omp_kernel(ni, nj, alpha, beta, C_outputFromOMP, A, B);

  /* Stop and print timer. */
  printf("OpenMP Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;
}

/* ------------------------------------------------------------- */
void compareResults(int ni, DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni),
                    DATA_TYPE POLYBENCH_2D(C_output, NI, NI, ni, ni)) {
  int i, j, fail;
  fail = 0;

  // Compare C with D
  for (i = 0; i < ni; i++) {
    for (j = 0; j < ni; j++) {
      // printf("%4.2f - %4.2f\n", C[i][j], C_output[i][j]);
      if (percentDiff(C[i][j], C_output[i][j]) >
          PERCENT_DIFF_ERROR_THRESHOLD) {
        fail++;
      }
    }
  }

  // print results
  printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f "
         "Percent: %d\n",
         PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

/* ------------------------------------------------------------- */
void GPU_argv_init() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice(GPU_DEVICE);
}

/* ------------------------------------------------------------- */
__global__ void syr2k_cuda_kernel(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
                             DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NI) && (j < NI)) {
    c[i * NI + j] *= beta;

    int k;
    for (k = 0; k < NJ; k++) {
      c[i * NI + j] += alpha * a[i * NJ + k] * b[j * NJ + k] +
                       alpha * b[i * NJ + k] * a[j * NJ + k];
    }
  }
}

/* ------------------------------------------------------------- */
void syr2k_cuda(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
               DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
               DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
               DATA_TYPE POLYBENCH_2D(C_inputToGpu, NI, NI, ni, ni),
               DATA_TYPE POLYBENCH_2D(C_outputFromGpu, NI, NI, ni, ni)) {

  GPU_argv_init();

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;
  DATA_TYPE *C_gpu;

  hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NI);
  hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
  hipMemcpy(C_gpu, C_inputToGpu, sizeof(DATA_TYPE) * NI * NI, hipMemcpyHostToDevice);

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((size_t)ceil(((float)NI) / ((float)DIM_THREAD_BLOCK_X)),
            (size_t)(ceil(((float)NI) / ((float)DIM_THREAD_BLOCK_Y))));

  /* Start timer. */
  polybench_start_instruments;

  syr2k_cuda_kernel<<<grid, block>>>(ni, nj, alpha, beta, A_gpu, B_gpu, C_gpu);
  hipDeviceSynchronize();

  /* Stop and print timer. */
  printf("GPU Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;

  hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NI,
             hipMemcpyDeviceToHost);

  hipFree(A_gpu);
  hipFree(B_gpu);
  hipFree(C_gpu);
}

/* ------------------------------------------------------------- */
/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni)) {
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < ni; j++) {
      fprintf(stderr, DATA_PRINTF_MODIFIER, C[i][j]);
      if ((i * ni + j) % 20 == 0)
        fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}

/* ------------------------------------------------------------- */
int main(int argc, char *argv[]) {
  /* Retrieve problem size. */
  int ni = NI;
  int nj = NJ;

  /* Variable declaration/allocation. */
  DATA_TYPE alpha;
  DATA_TYPE beta;
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(C, DATA_TYPE, NI, NI, ni, ni);
  POLYBENCH_2D_ARRAY_DECL(C_outputFromOMP, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(C_inputToGpu, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(C_outputFromGpu, DATA_TYPE, NI, NJ, ni, nj);

  init_arrays(ni, nj, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B),
              POLYBENCH_ARRAY(C));

  /*Copy the original C to C of OMP.*/
  // memcpy(C_outputFromOMP, C, sizeof(C_outputFromOMP));
  copy_array(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromOMP));
  
  *(C[0][0]) = (DATA_TYPE) 24;
  
  printf("%4.2f - %4.2f\n", *(C[0][0]), *(C_outputFromOMP[0][0]));

  // memcpy(C_inputToGpu, C, sizeof(C_inputToGpu));
  copy_array(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

  fprintf(stderr, "Calling Original.\n");
  syr2k_original(ni, nj, alpha, beta, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(A),
               POLYBENCH_ARRAY(B));

  fprintf(stderr, "Calling OMP.\n");
  syr2k_omp(ni, nj, alpha, beta, POLYBENCH_ARRAY(C_outputFromOMP), POLYBENCH_ARRAY(A),
               POLYBENCH_ARRAY(B));

  fprintf(stderr, "Calling compareResults(original, omp).\n");
  compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromOMP));

    fprintf(stderr, "Calling CUDA.\n");
  syr2k_cuda(ni, nj, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B),
            POLYBENCH_ARRAY(C_inputToGpu), POLYBENCH_ARRAY(C_outputFromGpu));

  fprintf(stderr, "Calling compareResults(original, cuda).\n");
  compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));
  
  polybench_prevent_dce(print_array(ni, POLYBENCH_ARRAY(C_outputFromGpu)));

  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);
  POLYBENCH_FREE_ARRAY(C);
  POLYBENCH_FREE_ARRAY(C_outputFromOMP);
  POLYBENCH_FREE_ARRAY(C_outputFromGpu);

  return 0;
}

#include <polybench.c>