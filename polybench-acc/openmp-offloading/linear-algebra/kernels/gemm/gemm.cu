#include "hip/hip_runtime.h"
/**
 * gemm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <hip/hip_runtime.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

#include <dlfcn.h>
#include <ffi.h>
// #include <stdio.h>
// #include <stdlib.h>
// #include <string.h>
#include <fcntl.h>
#include <stdint.h>
#include <inttypes.h>
#include <assert.h>

#define POLYBENCH_TIME 1

#include "gemm.h"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

#define GPU_DEVICE 0

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define RUN_ON_CPU

typedef struct Func {
  void *f;
  int nargs;
  ffi_type* arg_types[4];
  void* arg_values[4];
  ffi_type* ret_type;
  void* ret_value;
} Func;

/* Alternative Functions table pointer. */
Func ***table;

extern Func ***TablePointerFunctions;

/* current loop index. */
// extern long int current_loop_index;
long int current_loop_index;

bool create_target_functions_table(Func ****table_, int nrows, int ncolumns) {

  Func ***table;

  bool result = true;
  int i, j;

  fprintf(stderr, "Allocating the rows.\n");
  table = (Func ***) malloc(nrows * sizeof(Func **));

  if (table != NULL) {
    fprintf(stderr, "Allocating the columns.\n");

    for (i = 0; i < nrows; i++) {
      table[i] = (Func **) malloc(ncolumns * sizeof(Func *));
      if (table[i] != NULL) {
        for (j = 0; j < ncolumns; j++) {
          table[i][j] = (Func *) malloc(sizeof(Func));
        }
      } else {
        fprintf(stderr,
            "Error in table of target functions allocation (columns).\n");
        result = false;
      }
    }
  } else {
    fprintf(stderr,
        "Error in table of target functions allocation (rows).\n");
    result = false;
  }
  fprintf(stderr, "Allocating the columns is OK.\n");

  /*fprintf(stderr, "Initializing.\n");

  for (i = 0; i < nrows; i++) {
    for (j = 0; j < ncolumns; j++) {
      table[i][j][0] = 0;
    }
  }

  fprintf(stderr, "Initializing OK.\n");*/

  *table_ = table;

  return result;
}

/* Call the target function. */
void call_function_ffi_call(Func* ff) {
  fprintf(stderr," In call_function_ffi_call.\n");
  ffi_cif cif;

  if (ffi_prep_cif(&cif, FFI_DEFAULT_ABI, ff->nargs, ff->ret_type,
      ff->arg_types) != FFI_OK) {
    fprintf(stderr,"Error ffi_prep_cif.\n");
    exit(1);
  }

  ffi_call(&cif, FFI_FN(ff->f), ff->ret_value, ff->arg_values);
}

/* Arrays initialization. */
void init_array(int ni, int nj, int nk, DATA_TYPE *alpha, DATA_TYPE *beta,
                DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
                DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
                DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj)) {
  int i, j;

  *alpha = 32412;
  *beta = 2123;

  for (i = 0; i < ni; i++) {
    for (j = 0; j < nk; j++) {
      A[i][j] = ((DATA_TYPE)i * j) / NI;
    }
  }

  for (i = 0; i < nk; i++) {
    for (j = 0; j < nj; j++) {
      B[i][j] = ((DATA_TYPE)i * j) / NI;
    }
  }

  for (i = 0; i < ni; i++) {
    for (j = 0; j < nj; j++) {
      C[i][j] = ((DATA_TYPE)i * j) / NI;
    }
  }
}

/* Original Version. */
void gemm(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta,
          DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
          DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
          DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj)) {
  int i, j, k;

  for (i = 0; i < _PB_NI; i++) {
    for (j = 0; j < _PB_NJ; j++) {
      C[i][j] *= beta;

      for (k = 0; k < _PB_NK; ++k) {
        C[i][j] += alpha * A[i][k] * B[k][j];
      }
    }
  }
}

void gemm_original(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta,
          DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
          DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
          DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj)) {
  
  /* Start timer. */
  polybench_start_instruments;

  gemm(ni, nj, nk, alpha, beta, 
        A, 
        B,
        C);

  /* Stop and print timer. */
  printf("Original CPU Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
void gemm_omp_kernel(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta,
          DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
          DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
          DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj)) {

  int i, j, k;
  current_loop_index = 0;
  #pragma scop
  #pragma omp parallel
  {
  /* C := alpha*A*B + beta*C */
  #pragma omp for private(j, k) schedule(runtime)
    for (i = 0; i < _PB_NI; i++)
      for (j = 0; j < _PB_NJ; j++) {
        C[i][j] *= beta;
        for (k = 0; k < _PB_NK; ++k)
          C[i][j] += alpha * A[i][k] * B[k][j];
      }
  }
  #pragma endscop
}

void gemm_omp(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta,
              DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
              DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
              DATA_TYPE POLYBENCH_2D(C_outputFromOMP, NI, NJ, ni, nj)) {

  /* Start timer. */
  polybench_start_instruments;

  gemm_omp_kernel(ni, nj, nk, alpha, beta, 
                  A, 
                  B,
                  C_outputFromOMP);

  /* Stop and print timer. */
  printf("OpenMP Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;
}

/*--------------------------------------------------------------*/
/* CUDA */
void GPU_argv_init() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice(GPU_DEVICE);
}

__global__ void gemm_cuda_kernel(int ni, int nj, int nk, DATA_TYPE alpha,
                            DATA_TYPE beta, DATA_TYPE *a, DATA_TYPE *b,
                            DATA_TYPE *c) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < _PB_NI) && (j < _PB_NJ)) {
    c[i * NJ + j] *= beta;
    int k;
    for (k = 0; k < _PB_NK; k++) {
      c[i * NJ + j] += alpha * a[i * NK + k] * b[k * NJ + j];
    }
  }
}

void gemm_cuda(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta,
              DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
              DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
              DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(C_inputToGpu, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(C_outputFromGpu, NI, NJ, ni, nj)) {
  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;
  DATA_TYPE *C_gpu;

  fprintf(stderr, "Calling function gemm_cuda.\n");

  hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
  hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
  hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);

  hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
  hipMemcpy(C_gpu, C_inputToGpu, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((size_t)(ceil(((float)NI) / ((float)block.x))),
            (size_t)(ceil(((float)NJ) / ((float)block.y))));

  /* Start timer. */
  polybench_start_instruments;

  gemm_cuda_kernel<<<grid, block>>>(ni, nj, nk, alpha, beta, A_gpu, B_gpu, C_gpu);
  hipDeviceSynchronize();

  /* Stop and print timer. */
  printf("GPU Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;

  hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NJ,
             hipMemcpyDeviceToHost);

  hipFree(A_gpu);
  hipFree(B_gpu);
  hipFree(C_gpu);
}

void compareResults(int ni, int nj, DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj),
                    DATA_TYPE POLYBENCH_2D(C_outputFromGpu, NI, NJ, ni, nj)) {
  int i, j, fail;
  fail = 0;

  // Compare CPU and GPU outputs
  for (i = 0; i < ni; i++) {
    for (j = 0; j < nj; j++) {
      if (percentDiff(C[i][j], C_outputFromGpu[i][j]) >
          PERCENT_DIFF_ERROR_THRESHOLD) {
        fail++;
      }
    }
  }

  // Print results
  printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f "
         "Percent: %d\n",
         PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, int nj,
                        DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj)) {
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++) {
      fprintf(stderr, DATA_PRINTF_MODIFIER, C[i][j]);
      if ((i * ni + j) % 20 == 0)
        fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}

int main(int argc, char *argv[]) {
  /* Retrieve problem size. */
  int ni = NI;
  int nj = NJ;
  int nk = NK;

  /* Variable declaration/allocation. */
  DATA_TYPE alpha;
  DATA_TYPE beta;
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, NI, NK, ni, nk);
  POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, NK, NJ, nk, nj);
  POLYBENCH_2D_ARRAY_DECL(C, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(C_outputFromOMP, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(C_inputToGpu, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(C_outputFromGpu, DATA_TYPE, NI, NJ, ni, nj);


  init_array(ni, nj, nk, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B),
       POLYBENCH_ARRAY(C));

  /*Copy the original C to C of OMP.*/
  memcpy(C_outputFromOMP, C, sizeof(C_outputFromOMP));

  memcpy(C_inputToGpu, C, sizeof(C_inputToGpu));

  /* Preparing the call to target function.
  void gemm_cuda(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta,
              DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
              DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
              DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(C_inputToGpu, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(C_outputFromGpu, NI, NJ, ni, nj))
  */
  Func *ff = (Func *) malloc(sizeof(Func));

  ff->f = &gemm_cuda;
  memset(&ff->ret_value, 0, sizeof(ff->ret_value));

  // return type.
  ff->ret_type = &ffi_type_void;

  ff->nargs = 10;

  ff->arg_values[0] = &ni;
  ff->arg_values[1] = &nj;
  ff->arg_values[2] = &nk;
  ff->arg_values[3] = &alpha;
  ff->arg_values[4] = &beta;
  ff->arg_values[5] = &A;
  ff->arg_values[6] = &B;
  ff->arg_values[7] = &C;
  ff->arg_values[8] = &C_inputToGpu;
  ff->arg_values[9] = &C_outputFromGpu;
  ff->arg_values[10] = NULL;

  ff->arg_types[0] = &ffi_type_sint32;
  ff->arg_types[1] = &ffi_type_sint32;
  ff->arg_types[2] = &ffi_type_sint32;
  ff->arg_types[3] = &ffi_type_double;
  ff->arg_types[4] = &ffi_type_double;
  ff->arg_types[5] = &ffi_type_pointer;
  ff->arg_types[6] = &ffi_type_pointer;
  ff->arg_types[7] = &ffi_type_pointer;
  ff->arg_types[8] = &ffi_type_pointer;
  ff->arg_types[9] = &ffi_type_pointer;
  ff->arg_types[10] = NULL;

  int nloops = 1;
  int ndevices = 1;

  if (create_target_functions_table(&table, nloops, ndevices)) {
    // Set up the library Functions table.
    assert(table != NULL);

    fprintf(stderr, "Declaring function in 0,0.\n");
    table[0][0][0] = *ff;

    // TablePointerFunctions = table;
    // assert(TablePointerFunctions != NULL);
  }

  fprintf(stderr, "Calling gemm_original.\n");
  gemm_original(ni, nj, nk, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));

  fprintf(stderr, "Calling gemm_omp.\n");
  gemm_omp(ni, nj, nk, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C_outputFromOMP));

  fprintf(stderr, "Calling compareResults(original, omp).\n");
  compareResults(ni, nj, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromOMP));

  fprintf(stderr, "GPU init.\n");
  GPU_argv_init();

  fprintf(stderr, "Calling gemm_cuda.\n");
  gemm_cuda(ni, nj, nk, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_inputToGpu), POLYBENCH_ARRAY(C_outputFromGpu));

  // fprintf(stderr, "Calling using Table of Pointers 1.\n");
  // call_function_ffi_call(table[0][0]);

  fprintf(stderr, "Calling compareResults(original, cuda).\n");
  compareResults(ni, nj, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

  polybench_prevent_dce(print_array(ni, nj, POLYBENCH_ARRAY(C_outputFromGpu)));

  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);
  POLYBENCH_FREE_ARRAY(C);
  POLYBENCH_FREE_ARRAY(C_outputFromOMP);
  POLYBENCH_FREE_ARRAY(C_outputFromGpu);

  return 0;
}

#include <polybench.c>