#include "hip/hip_runtime.h"
/**
 * 3DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "convolution-3d.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

// Macros to generate openmp schedule.
#include <macros.h>

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0

#define RUN_ON_CPU

/* ------------------------------------------------------------- */
/* Array initialization. */
static void init_array(int ni, int nj, int nk,
                       DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk)) {
  int i, j, k;

  for (i = 0; i < ni; i++){
    for (j = 0; j < nj; j++){
      for (k = 0; k < nk; k++) {
        A[i][j][k] = i % 12 + 2 * (j % 7) + 3 * (k % 13);
      }
    }
  }
}

/* ------------------------------------------------------------- */
/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, int nj, int nk,
                        DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk)) {
  int i, j, k;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++)
      for (k = 0; k < nk; k++) {
        fprintf(stderr, DATA_PRINTF_MODIFIER, B[i][j][k]);
        if ((i * (nj * nk) + j * nk + k) % 20 == 0)
          fprintf(stderr, "\n");
      }
  fprintf(stderr, "\n");
}

/* ------------------------------------------------------------- */
void compareResults(int ni, int nj, int nk,
                    DATA_TYPE POLYBENCH_3D(B_ori, NI, NJ, NK, ni, nj, nk),
                    DATA_TYPE POLYBENCH_3D(B_out, NI, NJ, NK, ni, nj, nk)) {
  int i, j, k, fail;
  fail = 0;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++)
      for (k = 0; k < nk; k++) {
        if (percentDiff(B_ori[i][j][k], B_omp[i][j][k]) >
            PERCENT_DIFF_ERROR_THRESHOLD) {
          fail++;
        }
      }

  // Print results
  printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f "
         "Percent: %d\n",
         PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

/* ------------------------------------------------------------- */
void kernel_conv3d(int ni, int nj, int nk,
                          DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk),
                          DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk)) {
  int i, j, k;
  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

  c11 = +2;
  c21 = +5;
  c31 = -8;
  c12 = -3;
  c22 = +6;
  c32 = -9;
  c13 = +4;
  c23 = +7;
  c33 = +10;

  for (i = 1; i < _PB_NI - 1; ++i) // 0
  {
    for (j = 1; j < _PB_NJ - 1; ++j) // 1
    {
      for (k = 1; k < _PB_NK - 1; ++k) // 2
      {
        B[i][j][k] = c11 * A[(i - 1)][(j - 1)][(k - 1)] +
                     c13 * A[(i + 1)][(j - 1)][(k - 1)] +
                     c21 * A[(i - 1)][(j - 1)][(k - 1)] +
                     c23 * A[(i + 1)][(j - 1)][(k - 1)] +
                     c31 * A[(i - 1)][(j - 1)][(k - 1)] +
                     c33 * A[(i + 1)][(j - 1)][(k - 1)] +
                     c12 * A[(i + 0)][(j - 1)][(k + 0)] +
                     c22 * A[(i + 0)][(j + 0)][(k + 0)] +
                     c32 * A[(i + 0)][(j + 1)][(k + 0)] +
                     c11 * A[(i - 1)][(j - 1)][(k + 1)] +
                     c13 * A[(i + 1)][(j - 1)][(k + 1)] +
                     c21 * A[(i - 1)][(j + 0)][(k + 1)] +
                     c23 * A[(i + 1)][(j + 0)][(k + 1)] +
                     c31 * A[(i - 1)][(j + 1)][(k + 1)] +
                     c33 * A[(i + 1)][(j + 1)][(k + 1)];
      }
    }
  }
}

/* ------------------------------------------------------------- */
void conv3d_original(int ni, int nj, int nk,
                     DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk),
                     DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk)) {

  /* Start timer. */
  polybench_start_instruments;

  kernel_conv3d(ni, nj, nk, A, B);

  /* Stop and print timer. */
  polybench_stop_instruments;
  printf("CPU Time in seconds:\n");
  polybench_print_instruments;
}

/* ------------------------------------------------------------- */
/* Main computational kernel. The whole function will be timed,
         including the call and return. */
static void conv3d_omp_kernel(int ni, int nj, int nk,
                              DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk),
                              DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj,
                                                     nk)) {
  int i, j, k;
  #pragma scop
  #pragma omp parallel num_threads(OPENMP_NUM_THREADS)
  {
    #pragma omp for private(j, k) collapse(2) schedule(OPENMP_SCHEDULE_WITH_CHUNK)
    for (i = 1; i < _PB_NI - 1; ++i)
      for (j = 1; j < _PB_NJ - 1; ++j)
        for (k = 1; k < _PB_NK - 1; ++k) {
          B[i][j][k] =
              2 * A[i - 1][j - 1][k - 1] + 4 * A[i + 1][j - 1][k - 1] +
              5 * A[i - 1][j - 1][k - 1] + 7 * A[i + 1][j - 1][k - 1] +
              -8 * A[i - 1][j - 1][k - 1] + 10 * A[i + 1][j - 1][k - 1] +
              -3 * A[i][j - 1][k] + 6 * A[i][j][k] + -9 * A[i][j + 1][k] +
              2 * A[i - 1][j - 1][k + 1] + 4 * A[i + 1][j - 1][k + 1] +
              5 * A[i - 1][j][k + 1] + 7 * A[i + 1][j][k + 1] +
              -8 * A[i - 1][j + 1][k + 1] + 10 * A[i + 1][j + 1][k + 1];
        }
  }
  #pragma endscop
}

/* ------------------------------------------------------------- */
void conv3d_omp(int ni, int nj, int nk,
                DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk),
                DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk)) {

  /* Start timer. */
  polybench_start_instruments;

  conv3d_omp_kernel(ni, nj, nk, A, B);

  /* Stop and print timer. */
  polybench_stop_instruments;
  printf("CPU-OMP Time in seconds:\n");
  polybench_print_instruments;
}

/* ------------------------------------------------------------- */
void GPU_argv_init() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice(GPU_DEVICE);
}

/* ------------------------------------------------------------- */
__global__ void conv3d_cuda_kernel(int ni, int nj, int nk, DATA_TYPE *A,
                                     DATA_TYPE *B, int i) {
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

  c11 = +2;
  c21 = +5;
  c31 = -8;
  c12 = -3;
  c22 = +6;
  c32 = -9;
  c13 = +4;
  c23 = +7;
  c33 = +10;

  if ((i < (_PB_NI - 1)) && (j < (_PB_NJ - 1)) && (k < (_PB_NK - 1)) &&
      (i > 0) && (j > 0) && (k > 0)) {
    B[i * (NK * NJ) + j * NK + k] =
        c11 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c21 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c23 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c31 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c33 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k - 1)] +
        c12 * A[(i + 0) * (NK * NJ) + (j - 1) * NK + (k + 0)] +
        c22 * A[(i + 0) * (NK * NJ) + (j + 0) * NK + (k + 0)] +
        c32 * A[(i + 0) * (NK * NJ) + (j + 1) * NK + (k + 0)] +
        c11 * A[(i - 1) * (NK * NJ) + (j - 1) * NK + (k + 1)] +
        c13 * A[(i + 1) * (NK * NJ) + (j - 1) * NK + (k + 1)] +
        c21 * A[(i - 1) * (NK * NJ) + (j + 0) * NK + (k + 1)] +
        c23 * A[(i + 1) * (NK * NJ) + (j + 0) * NK + (k + 1)] +
        c31 * A[(i - 1) * (NK * NJ) + (j + 1) * NK + (k + 1)] +
        c33 * A[(i + 1) * (NK * NJ) + (j + 1) * NK + (k + 1)];
  }
}

/* ------------------------------------------------------------- */
void conv3d_cuda(int ni, int nj, int nk,
                       DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk),
                       DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk),
                       DATA_TYPE POLYBENCH_3D(B_outputFromGpu, NI, NJ, NK, ni,
                                              nj, nk)) {
  fprintf(stderr, "Calling function conv3d_cuda.\n");

  // GPU initialization.
  GPU_argv_init();

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;

  hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
  hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
  hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ * NK,
             hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NI * NJ * NK,
             hipMemcpyHostToDevice);

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((size_t)(ceil(((float)NK) / ((float)block.x))),
            (size_t)(ceil(((float)NJ) / ((float)block.y))));

  /* Start timer. */
  // polybench_start_instruments;

  int i;
  for (i = 1; i < _PB_NI - 1; ++i) // 0
  {
    convolution3D_kernel<<<grid, block>>>(ni, nj, nk, A_gpu, B_gpu, i);
  }

  hipDeviceSynchronize();
  // printf("GPU Time in seconds:\n");
  // polybench_stop_instruments;
  // polybench_print_instruments;

  hipMemcpy(B_outputFromGpu, B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK,
             hipMemcpyDeviceToHost);

  hipFree(A_gpu);
  hipFree(B_gpu);
}

/* ------------------------------------------------------------- */
int main(int argc, char *argv[]) {
  int ni = NI;
  int nj = NJ;
  int nk = NK;

  POLYBENCH_3D_ARRAY_DECL(A, DATA_TYPE, NI, NJ, NK, ni, nj, nk);
  POLYBENCH_3D_ARRAY_DECL(B, DATA_TYPE, NI, NJ, NK, ni, nj, nk);
  POLYBENCH_3D_ARRAY_DECL(B_outputFromOMP, DATA_TYPE, NI, NJ, NK, ni, nj, nk);
  POLYBENCH_3D_ARRAY_DECL(B_outputFromGpu, DATA_TYPE, NI, NJ, NK, ni, nj, nk);

  fprintf(stderr, "Preparing alternatives functions.\n");
  /* Preparing the call to target function.
  void conv3d_cuda(int ni, int nj, int nk,
                       DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk),
                       DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk),
                       DATA_TYPE POLYBENCH_3D(B_outputFromGpu, NI, NJ, NK, ni, nj, nk))
  */
  // Number of parameters to function.
  int n_params = 6;

  // void handler_function_init_array_GPU(void)
  Func *ff_0 = (Func *)malloc(sizeof(Func));

  // Number of arguments + 1, the lists need to have last element NULL.
  ff_0->arg_types = (ffi_type **)malloc((n_params + 1) * sizeof(ffi_type *));
  ff_0->arg_values = (void **)malloc((n_params + 1) * sizeof(void *));

  ff_0->f = &conv3d_cuda;
  memset(&ff_0->ret_value, 0, sizeof(ff_0->ret_value));

  // return type.
  ff_0->ret_type = &ffi_type_void;

  ff_0->nargs = n_params;

  ff_0->arg_values[0] = &ni;
  ff_0->arg_values[1] = &nj;
  ff_0->arg_values[2] = &nk;
  ff_0->arg_values[3] = &A;
  ff_0->arg_values[4] = &B;
  ff_0->arg_values[5] = &B_outputFromGpu;
  ff_0->arg_values[6] = NULL;

  ff_0->arg_types[0] = &ffi_type_sint32;
  ff_0->arg_types[1] = &ffi_type_sint32;
  ff_0->arg_types[2] = &ffi_type_sint32;
  ff_0->arg_types[3] = &ffi_type_pointer;
  ff_0->arg_types[4] = &ffi_type_pointer;
  ff_0->arg_types[5] = &ffi_type_pointer;
  ff_0->arg_types[6] = NULL;

  /*          device 0
   * loop 0   conv3d_cuda
   * matrix 1 x 1.
  */
  fprintf(stderr, "Creating table of target functions.\n");
  int nloops = 1;
  int ndevices = 2;

  if (create_target_functions_table(&table, nloops, ndevices)) {
    // Set up the library Functions table.
    assert(table != NULL);
    // 0,0 is CPU = NULL, because is openmp code.
    fprintf(stderr, "Declaring function in 0,1.\n");
    table[0][1][0] = *ff_0;

    TablePointerFunctions = table;
    assert(TablePointerFunctions != NULL);
  }

  fprintf(stderr, "Calling init_array.\n");
  init_array(ni, nj, nk, POLYBENCH_ARRAY(A));

  // convolution3DCuda(ni, nj, nk, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_outputFromGpu));

  fprintf(stdout, "exp, num_threads, NI, NJ, NK, ORIG, OMP\n");

  fprintf(stdout, "OMP+OFF, %d, %d, %d, %d, ", OPENMP_NUM_THREADS, NI, NJ, NK);

  fprintf(stderr, "Calling conv3d_original.\n");
  conv3d_original(ni, nj, nk, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));
  fprintf(stdout, ", ");

  fprintf(stderr, "Calling conv3d_omp.\n");
  conv3d_omp(ni, nj, nk, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B_OMP));
  fprintf(stdout, "\n");

  fprintf(stderr, "Calling compareResults(original, omp).\n");
  compareResults(ni, nj, nk, POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_outputFromOMP));

  fprintf(stderr, "Calling compareResults(original, cuda).\n");
  compareResults(ni, nj, nk, POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_outputFromGpu));

  polybench_prevent_dce(print_array(NI, NJ, NK, POLYBENCH_ARRAY(B_outputFromGpu)));

  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);
  POLYBENCH_FREE_ARRAY(B_outputFromOMP);
  POLYBENCH_FREE_ARRAY(B_outputFromGpu);

  return 0;
}

// polybench.c uses the OpenMP to parallelize somethings. This call were
// intercepted by hookomp.
#undef _OPENMP

#include <polybench.c>
