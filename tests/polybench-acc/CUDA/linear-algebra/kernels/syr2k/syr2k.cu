#include "hip/hip_runtime.h"
/**
 * syr2k.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "syr2k.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

// Time measures implementation.
#include <timing.h>

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

#define RUN_ON_CPU

/* ------------------------------------------------------------- */
/* Arrays initialization. */
void init_arrays(int ni, int nj,
		DATA_TYPE *alpha,
		DATA_TYPE *beta,
		DATA_TYPE POLYBENCH_2D(A,NI,NJ,ni,nj),
		DATA_TYPE POLYBENCH_2D(B,NI,NJ,ni,nj),
		DATA_TYPE POLYBENCH_2D(C,NI,NI,ni,ni))
{
  int i, j;

  *alpha = 32412;
  *beta = 2123;

  for (i = 0; i < ni; i++) {
    for (j = 0; j < nj; j++) {
      A[i][j] = ((DATA_TYPE)i * j) / ni;
      B[i][j] = ((DATA_TYPE)i * j) / ni;
    }
  }

  for (i = 0; i < ni; i++) {
    for (j = 0; j < ni; j++) {
      C[i][j] = ((DATA_TYPE)i * j) / ni;
    }
  }
}

/* ------------------------------------------------------------- */
void copy_array(int ni, int nj, DATA_TYPE POLYBENCH_2D(C_source, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(C_dest, NI, NJ, ni, nj)) {
  int i, j;

  for (i = 0; i < ni; i++) {
    for (j = 0; j < nj; j++) {
      C_dest[i][j] = C_source[i][j];
      // printf("%4.2f - %4.2f\n", C_dest[i][j], C_source[i][j]);
    }
  }
}

/* ------------------------------------------------------------- */
/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int ni, DATA_TYPE POLYBENCH_2D(C,NI,NI,ni,ni)) {
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < ni; j++) {
      fprintf(stderr, DATA_PRINTF_MODIFIER, C[i][j]);
      if ((i * ni + j) % 20 == 0) fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}

/* ------------------------------------------------------------- */
/* Original Version. */
void syr2kCpu(int ni, int nj,
		DATA_TYPE alpha,
		DATA_TYPE beta,
		DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
		DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
		DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni)) {
  int i, j, k;

  /*    C := alpha*A*B' + alpha*B*A' + beta*C */
  for (i = 0; i < _PB_NI; i++) {
    for (j = 0; j < _PB_NI; j++) {
      C[i][j] *= beta;
    }
  }

  for (i = 0; i < _PB_NI; i++) {
    for (j = 0; j < _PB_NI; j++) {
      for (k = 0; k < _PB_NJ; k++) {
        C[i][j] += alpha * A[i][k] * B[j][k];
        C[i][j] += alpha * B[i][k] * A[j][k];
      }
    }
  }
}

/* ------------------------------------------------------------- */
void syr2k_original(int ni, int nj,
			DATA_TYPE alpha,
			DATA_TYPE beta,
			DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
			DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
			DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni)) {

  /* Start timer. */
  // polybench_start_instruments;
  HOOKOMP_TIMING_SEQ_START;

  /* Run kernel. */
  syr2kCpu(ni, nj, alpha, beta, A, B, C);

  /* Stop and print timer. */
  // polybench_stop_instruments;
  // // printf("Original CPU Time in seconds:\n");
  // polybench_print_instruments;
  HOOKOMP_TIMING_SEQ_STOP;
  // HOOKOMP_TIMING_SEQ_PRINT;
}

/* ------------------------------------------------------------- */
void GPU_argv_init() {
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
	hipSetDevice(GPU_DEVICE);
}

/* ------------------------------------------------------------- */
void compareResults(int ni,
			DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni),
			DATA_TYPE POLYBENCH_2D(C_outputFromGpu,
			NI, NI, ni, ni)) {
  int i, j, fail;
  fail = 0;

  // Compare C with D
  for (i = 0; i < ni; i++) {
    for (j = 0; j < ni; j++) {
      if (percentDiff(C[i][j], C_outputFromGpu[i][j]) > 
        PERCENT_DIFF_ERROR_THRESHOLD) {
        fail++;
      }
    }
  }

  // print results
  fprintf(stderr,
          "Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}
/* ------------------------------------------------------------- */
__global__ void syr2k_cuda_kernel(int ni, int nj,
				  DATA_TYPE alpha,
				  DATA_TYPE beta,
				  DATA_TYPE *a,
				  DATA_TYPE *b,
				  DATA_TYPE *c) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NI) && (j < NI)) {
    c[i * NI + j] *= beta;

    int k;
    for (k = 0; k < NJ; k++) {
      c[i * NI + j] += alpha * a[i * NJ + k] * b[j * NJ + k] +
                     alpha * b[i * NJ + k] * a[j * NJ + k];
    }
  }
}
/* ------------------------------------------------------------- */
__global__ void syr2k_cuda_kernel_0(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
                             DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NI) && (j < NI)) {
    c[i * NI + j] *= beta;
  }
}

/* ------------------------------------------------------------- */
void syr2k_cuda(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta, 
				DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj), 
				DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
				DATA_TYPE POLYBENCH_2D(C_inputToGpu, NI, NI, ni, ni), 
				DATA_TYPE POLYBENCH_2D(C_outputFromGpu, NI, NI, ni, ni)) 
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NI);

	HOOKOMP_TIMING_DT_H2D_START;

	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C_inputToGpu, sizeof(DATA_TYPE) * NI * NI, hipMemcpyHostToDevice);

	HOOKOMP_TIMING_DT_H2D_STOP;
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)ceil( ((float)NI) / ((float)DIM_THREAD_BLOCK_X) ), (size_t)(ceil( ((float)NI) / ((float)DIM_THREAD_BLOCK_Y) )));
	
	/* Start timer. */
  	// polybench_start_instruments;
  	HOOKOMP_TIMING_DEV_START;

	syr2k_cuda_kernel<<<grid,block>>>(ni, nj, alpha, beta, A_gpu, B_gpu, C_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	// printf("GPU Time in seconds:\n");
  	// polybench_stop_instruments;
 	// polybench_print_instruments;

 	HOOKOMP_TIMING_DEV_STOP;

 	HOOKOMP_TIMING_DT_D2H_START;
		
	hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NI, hipMemcpyDeviceToHost);

	HOOKOMP_TIMING_DT_D2H_STOP;

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
}

/* ------------------------------------------------------------- */
int main(int argc, char *argv[])
{
	/* Retrieve problem size. */
	int ni = NI;
	int nj = NJ;

	/* Variable declaration/allocation. */
	DATA_TYPE alpha;
	DATA_TYPE beta;
	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(C,DATA_TYPE,NI,NI,ni,ni);
	POLYBENCH_2D_ARRAY_DECL(C_inputToGpu, DATA_TYPE, NI, NI, ni, ni);
  	POLYBENCH_2D_ARRAY_DECL(C_outputFromGpu, DATA_TYPE, NI, NI, ni, ni);

  	fprintf(stderr, "Calling init_array.\n");
	init_arrays(ni, nj, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));

	fprintf(stderr, "Copying C to C_outputFromGpu.\n");
  	copy_array(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

  	fprintf(stderr, "Calling syr2k_original:\n");
  	syr2k_original(ni, nj, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));

  	fprintf(stderr, "Calling syr2k_cuda:\n");
	GPU_argv_init();
	syr2k_cuda(ni, nj, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C_inputToGpu), POLYBENCH_ARRAY(C_outputFromGpu));

	fprintf(stdout, "version = CUDA, num_threads = %d, NI = %d, NJ = %d, NK = %d, ", 1, NI, NJ, 0);
	HOOKOMP_PRINT_TIME_RESULTS;

	fprintf(stderr, "Calling compareResults(original, cuda).\n");
  	compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

  	polybench_prevent_dce(print_array(ni, POLYBENCH_ARRAY(C_outputFromGpu)));

	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(B);
	POLYBENCH_FREE_ARRAY(C);
	POLYBENCH_FREE_ARRAY(C_inputToGpu);
	POLYBENCH_FREE_ARRAY(C_outputFromGpu);

  return 0;
}

// polybench.c uses the OpenMP to parallelize somethings. This call were
// intercepted by hookomp.
#undef _OPENMP

#include <polybench.c>