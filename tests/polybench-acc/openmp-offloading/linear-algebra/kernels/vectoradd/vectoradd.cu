#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

#define POLYBENCH_TIME 1

#include "vectoradd.h"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

// Macros to generate openmp schedule.
#include <macros.h>

// Time measures implementation.
#include <timing.h>

// Offloading support functions.
#include <offload.h>

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

#define RUN_ON_CPU

/* ------------------------------------------------------------- */
/* Arrays initialization. */
void init_array(int ni, DATA_TYPE POLYBENCH_1D(A, NI, ni),
                DATA_TYPE POLYBENCH_1D(B, NI, ni),
                DATA_TYPE POLYBENCH_1D(C, NI, ni)) {
  int i;

  for (i = 0; i < ni; i++) {
      A[i] = ((DATA_TYPE) i ) / NI;
  }

  for (i = 0; i < ni; i++) {
      B[i] = ((DATA_TYPE) i ) / NI;
  }

  for (i = 0; i < ni; i++) {
      C[i] = 0.0;
  }
}

/* ------------------------------------------------------------- */
void copy_array(int ni, DATA_TYPE POLYBENCH_1D(C_source, NI, ni), DATA_TYPE POLYBENCH_1D(C_dest, NI, ni)) {
  int i;

  for (i = 0; i < ni; i++) {
      C_dest[i] = C_source[i];
      // printf("%4.2f - %4.2f\n", C_dest[i][j], C_source[i][j]);
  }
}

/* ------------------------------------------------------------- */
void compareResults(int ni, DATA_TYPE POLYBENCH_1D(C, NI, ni),
                    DATA_TYPE POLYBENCH_1D(C_output, NI, ni)) {
  int i, fail;
  fail = 0;

  // Compare CPU and GPU outputs
  for (i = 0; i < ni; i++) {
      if (percentDiff(C[i], C_output[i]) > PERCENT_DIFF_ERROR_THRESHOLD) {
        fail++;
      }
  }

  // Print results
  fprintf(stderr,
          "Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f "
          "Percent: %d\n",
          PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

/* ------------------------------------------------------------- */
/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni,
                        DATA_TYPE POLYBENCH_1D(C, NI, ni)) {
  int i;

  for (i = 0; i < ni; i++){
    fprintf(stderr, DATA_PRINTF_MODIFIER, C[i]);
  }
  fprintf(stderr, "\n");
}

/* ------------------------------------------------------------- */
/* Original Version. */
void vectoradd(int ni, DATA_TYPE POLYBENCH_1D(A, NI, ni),
          DATA_TYPE POLYBENCH_1D(B, NI, ni),
          DATA_TYPE POLYBENCH_1D(C, NI, ni)) {
  int i;

  for (i = 0; i < _PB_NI; i++) {
        C[i] = A[i] + B[i];
  }
}

/* ------------------------------------------------------------- */
void vectoradd_original(int ni, DATA_TYPE POLYBENCH_1D(A, NI, ni),
                   DATA_TYPE POLYBENCH_1D(B, NI, ni),
                   DATA_TYPE POLYBENCH_1D(C, NI, ni)) {

  /* Start timer. */
  HOOKOMP_TIMING_SEQ_START;

  vectoradd(ni, A, B, C);

  /* Stop and print timer. */
  HOOKOMP_TIMING_SEQ_STOP;
}

/* ------------------------------------------------------------- */
/* Main computational kernel. The whole function will be timed,
   including the call and return. */
void vectoradd_omp_kernel(int ni, DATA_TYPE POLYBENCH_1D(A, NI, ni),
                     DATA_TYPE POLYBENCH_1D(B, NI, ni),
                     DATA_TYPE POLYBENCH_1D(C, NI, ni)) {

  int i;

  #pragma scop
  #pragma omp parallel num_threads(OPENMP_NUM_THREADS)
  {
    /* C := alpha*A*B + beta*C */
    current_loop_index = 0;
    num_threads_defined = OPENMP_NUM_THREADS;
    // Copy to device A, B, C.
    q_data_transfer_write = (sizeof(DATA_TYPE) * NI) +
                            (sizeof(DATA_TYPE) * NI);
    // Copy back C.
    q_data_transfer_read = (sizeof(DATA_TYPE) * NI);

    // 0: MEMORY_ALLOC_DEFAULT, 1: MEMORY_ALLOC_PAGEABLE, 2: MEMORY_ALLOC_PINNED
    type_of_data_allocation = MEMORY_ALLOC_PAGEABLE;
    #pragma omp for private(i) schedule(OPENMP_SCHEDULE_WITH_CHUNK)
    for (i = 0; i < _PB_NI; i++) {
      C[i] = A[i] + B[i];
    }
  }
  #pragma endscop
}
/* ------------------------------------------------------------- */
void vectoradd_omp(int ni, DATA_TYPE POLYBENCH_1D(A, NI, ni),
              DATA_TYPE POLYBENCH_1D(B, NI, ni),
              DATA_TYPE POLYBENCH_1D(C_outputFromOMP, NI, ni)) {

  /* Start timer. */
  HOOKOMP_TIMING_OMP_START;

  vectoradd_omp_kernel(ni, A, B, C_outputFromOMP);

  /* Stop and print timer. */
  HOOKOMP_TIMING_OMP_STOP;
}

/*--------------------------------------------------------------*/
/* CUDA */
void GPU_argv_init() {
  hipDeviceProp_t deviceProp;
  fprintf(stderr, "GPU init.\n");

  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  fprintf(stderr, "setting device %d with name %s\n", GPU_DEVICE,
          deviceProp.name);
  hipSetDevice(GPU_DEVICE);
}

/* ------------------------------------------------------------- */
__global__ void vectoradd_cuda_kernel(int ni, DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < _PB_NI) {
    c[i] = a[i] + b[i];
  }
}

/* ------------------------------------------------------------- */
void vectoradd_cuda(int ni, DATA_TYPE POLYBENCH_1D(A, NI, ni),
               DATA_TYPE POLYBENCH_1D(B, NI, ni),
               DATA_TYPE POLYBENCH_1D(C, NI, ni),
               DATA_TYPE POLYBENCH_1D(C_inputToGpu, NI, ni),
               DATA_TYPE POLYBENCH_1D(C_outputFromGpu, NI, ni)) {

  fprintf(stderr, "Calling function vectoradd_cuda.\n");

  // GPU initialization.
  GPU_argv_init();

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;
  DATA_TYPE *C_gpu;

  hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI);
  hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI);
  hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI);

  HOOKOMP_TIMING_DT_H2D_START;

  hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI, hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NI, hipMemcpyHostToDevice);
  hipMemcpy(C_gpu, C_inputToGpu, sizeof(DATA_TYPE) * NI, hipMemcpyHostToDevice);

  HOOKOMP_TIMING_DT_H2D_STOP;

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((unsigned int)ceil( ((float)N) / ((float)block.x) ), 1);

  /* Start timer. */
  HOOKOMP_TIMING_DEV_KERNEL1_START;

  vectoradd_cuda_kernel<<<grid, block>>>(ni, A_gpu, B_gpu, C_gpu);
  hipDeviceSynchronize();

  /* Stop and print timer. */
  HOOKOMP_TIMING_DEV_KERNEL1_STOP;

  HOOKOMP_TIMING_DT_D2H_START;

  hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI, hipMemcpyDeviceToHost);

  HOOKOMP_TIMING_DT_D2H_STOP;

  hipFree(A_gpu);
  hipFree(B_gpu);
  hipFree(C_gpu);
}

/* ------------------------------------------------------------- */
int main(int argc, char *argv[]) {
  /* Retrieve problem size. */
  int ni = NI;
  
  /* Variable declaration/allocation. */
  POLYBENCH_1D_ARRAY_DECL(A, DATA_TYPE, NI, ni);
  POLYBENCH_1D_ARRAY_DECL(B, DATA_TYPE, NI, ni);
  POLYBENCH_1D_ARRAY_DECL(C, DATA_TYPE, NI, ni);
  POLYBENCH_1D_ARRAY_DECL(C_outputFromOMP, DATA_TYPE, NI, ni);
  POLYBENCH_1D_ARRAY_DECL(C_inputToGpu, DATA_TYPE, NI, ni);
  POLYBENCH_1D_ARRAY_DECL(C_outputFromGpu, DATA_TYPE, NI, ni);

  fprintf(stderr, "Preparing alternatives functions.\n");
  /* Preparing the call to target function.
  void vectoradd_cuda(int ni, DATA_TYPE POLYBENCH_1D(A, NI, ni),
               DATA_TYPE POLYBENCH_1D(B, NI, ni),
               DATA_TYPE POLYBENCH_1D(C, NI, ni),
               DATA_TYPE POLYBENCH_1D(C_inputToGpu, NI, ni),
               DATA_TYPE POLYBENCH_1D(C_outputFromGpu, NI, ni))
  */
  // Number of parameters to function.
  int n_params = 6;

  // void handler_function_init_array_GPU(void)
  Func *ff_0 = (Func *)malloc(sizeof(Func));

  // Number of arguments + 1, the lists need to have last element NULL.
  ff_0->arg_types = (ffi_type **)malloc((n_params + 1) * sizeof(ffi_type *));
  ff_0->arg_values = (void **)malloc((n_params + 1) * sizeof(void *));

  ff_0->f = &vectoradd_cuda;
  memset(&ff_0->ret_value, 0, sizeof(ff_0->ret_value));

  // return type.
  ff_0->ret_type = &ffi_type_void;

  ff_0->nargs = n_params;

  ff_0->arg_values[0] = &ni;
  ff_0->arg_values[1] = &A;
  ff_0->arg_values[2] = &B;
  ff_0->arg_values[3] = &C;
  ff_0->arg_values[4] = &C_inputToGpu;
  ff_0->arg_values[5] = &C_outputFromGpu;
  ff_0->arg_values[6] = NULL;

  ff_0->arg_types[0] = &ffi_type_sint32;
  ff_0->arg_types[1] = &ffi_type_pointer;
  ff_0->arg_types[2] = &ffi_type_pointer;
  ff_0->arg_types[3] = &ffi_type_pointer;
  ff_0->arg_types[4] = &ffi_type_pointer;
  ff_0->arg_types[5] = &ffi_type_pointer;
  ff_0->arg_types[6] = NULL;

  /*          device 0
   * loop 0   vectoradd_cuda
   * matrix 1 x 1.
  */
  fprintf(stderr, "Creating table of target functions.\n");
  int nloops = 1;
  int ndevices = 2;

  if (create_target_functions_table(&table, nloops, ndevices)) {
    // Set up the library Functions table.
    assert(table != NULL);
    // 0,0 is CPU = NULL, because is openmp code.
    fprintf(stderr, "Declaring function in 0,1.\n");
    table[0][1][0] = *ff_0;

    TablePointerFunctions = table;
    assert(TablePointerFunctions != NULL);
  }

  fprintf(stderr, "Calling init_array.\n");
  init_array(ni, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));

  /*Copy the original C to C of OMP.*/
  copy_array(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromOMP));

  copy_array(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

#ifdef RUN_ORIG_VERSION
  fprintf(stderr, "Calling vectoradd_original:\n");
  vectoradd_original(ni, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));
#endif

  fprintf(stderr, "Calling vectoradd_omp:\n");
  vectoradd_omp(ni, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C_outputFromOMP));

  fprintf(stdout, "version = OMP+OFF, num_threads = %d, NI = %d, ", OPENMP_NUM_THREADS, NI);
  HOOKOMP_PRINT_TIME_RESULTS;

  fprintf(stderr, "Calling compareResults(original, omp).\n");
  compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromOMP));

  fprintf(stderr, "Calling compareResults(original, cuda).\n");
  compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

  polybench_prevent_dce(print_array(ni, POLYBENCH_ARRAY(C_outputFromGpu)));

  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);
  POLYBENCH_FREE_ARRAY(C);
  POLYBENCH_FREE_ARRAY(C_outputFromOMP);
  POLYBENCH_FREE_ARRAY(C_outputFromGpu);

  return 0;
}

// polybench.c uses the OpenMP to parallelize somethings. This call were
// intercepted by hookomp.
#undef _OPENMP

#include <polybench.c>
