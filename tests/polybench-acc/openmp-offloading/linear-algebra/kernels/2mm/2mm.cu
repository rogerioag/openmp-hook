#include "hip/hip_runtime.h"
/**
 * 2mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

#define POLYBENCH_TIME 1

#include "2mm.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

// Offloading support functions.
#include <offload.h>

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

#define RUN_ON_CPU

/* GPU pointers now as global to be shared between kernels. */
DATA_TYPE *tmp_gpu;
DATA_TYPE *A_gpu;
DATA_TYPE *B_gpu;
DATA_TYPE *C_gpu;
DATA_TYPE *D_gpu;

// If data pointer was allocated in GPU Memory.
bool gpu_data_allocated = false;
bool gpu_data_copied = false;

/* ------------------------------------------------------------- */
/* Arrays initialization. */
void init_array(int ni, int nj, int nk, int nl, DATA_TYPE *alpha,
                DATA_TYPE *beta, DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
                DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
                DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj),
                DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl)) {
  int i, j;

  *alpha = 32412;
  *beta = 2123;

  for (i = 0; i < ni; i++) {
    for (j = 0; j < nk; j++) {
      A[i][j] = ((DATA_TYPE)i * j) / NI;
    }
  }

  for (i = 0; i < nk; i++) {
    for (j = 0; j < nj; j++) {
      B[i][j] = ((DATA_TYPE)i * (j + 1)) / NJ;
    }
  }

  for (i = 0; i < nl; i++) {
    for (j = 0; j < nj; j++) {
      C[i][j] = ((DATA_TYPE)i * (j + 3)) / NL;
    }
  }

  for (i = 0; i < ni; i++) {
    for (j = 0; j < nl; j++) {
      D[i][j] = ((DATA_TYPE)i * (j + 2)) / NK;
    }
  }
}

/* ------------------------------------------------------------- */
void compareResults(int ni, int nl, DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl),
                    DATA_TYPE POLYBENCH_2D(D_outputFromGpu, NI, NL, ni, nl)) {
  int i, j, fail;
  fail = 0;

  for (i = 0; i < ni; i++) {
    for (j = 0; j < nl; j++) {
      if (percentDiff(D[i][j], D_outputFromGpu[i][j]) >
          PERCENT_DIFF_ERROR_THRESHOLD) {
        fail++;
      }
    }
  }

  // print results
  printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f "
         "Percent: %d\n",
         PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

/* ------------------------------------------------------------- */
/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, int nl,
                        DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl)) {
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nl; j++) {
      fprintf(stderr, DATA_PRINTF_MODIFIER, D[i][j]);
      if ((i * ni + j) % 20 == 0)
        fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}

/* ------------------------------------------------------------- */
/* Original kernel. */
void mm2_cpu(int ni, int nj, int nk, int nl, DATA_TYPE alpha, DATA_TYPE beta,
             DATA_TYPE POLYBENCH_2D(tmp, NI, NJ, ni, nj),
             DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
             DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
             DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj),
             DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl)) {
  int i, j, k;

  /* D := alpha*A*B*C + beta*D */
  for (i = 0; i < _PB_NI; i++) {
    for (j = 0; j < _PB_NJ; j++) {
      tmp[i][j] = 0;
      for (k = 0; k < _PB_NK; ++k) {
        tmp[i][j] += alpha * A[i][k] * B[k][j];
      }
    }
  }

  for (i = 0; i < _PB_NI; i++) {
    for (j = 0; j < _PB_NL; j++) {
      D[i][j] *= beta;
      for (k = 0; k < _PB_NJ; ++k) {
        D[i][j] += tmp[i][k] * C[k][j];
      }
    }
  }
}

/* ------------------------------------------------------------- */
/* Original Version. */
void mm_original(int ni, int nj, int nk, int nl, DATA_TYPE alpha, DATA_TYPE beta,
             DATA_TYPE POLYBENCH_2D(tmp, NI, NJ, ni, nj),
             DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
             DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
             DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj),
             DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl)) {
  
  /* Start timer. */
  polybench_start_instruments;

  mm2_cpu(ni, nj, nk, nl,
	      alpha, beta,
	      tmp,
	      A,
	      B,
	      C,
	      D);

  /* Stop and print timer. */
  printf("Original CPU Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;
}

/* ------------------------------------------------------------- */
/* OMP Version. */
/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_2mm(int ni, int nj, int nk, int nl,
		DATA_TYPE alpha,
		DATA_TYPE beta,
		DATA_TYPE POLYBENCH_2D(tmp,NI,NJ,ni,nj),
		DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
		DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
		DATA_TYPE POLYBENCH_2D(C,NL,NJ,nl,nj),
		DATA_TYPE POLYBENCH_2D(D,NI,NL,ni,nl))
{
  int i, j, k;
  #pragma scop
  /* D := alpha*A*B*C + beta*D */
  #pragma omp parallel
  {
  	current_loop_index = 0;
    #pragma omp for private(j, k) schedule(runtime)
    for (i = 0; i < _PB_NI; i++){
      for (j = 0; j < _PB_NJ; j++){
    	tmp[i][j] = 0;
  	  	for (k = 0; k < _PB_NK; ++k){
	    	tmp[i][j] += alpha * A[i][k] * B[k][j];
        }
      }
    }
    current_loop_index = 1;
    #pragma omp for private(j, k) schedule(runtime)
    for (i = 0; i < _PB_NI; i++){
      for (j = 0; j < _PB_NL; j++){
	  	D[i][j] *= beta;
	  	for (k = 0; k < _PB_NJ; ++k){
	    	D[i][j] += tmp[i][k] * C[k][j];
	    }
	  }
	}
  }
  #pragma endscop
}

/* ------------------------------------------------------------- */
/* OMP Version. */
void mm_omp(int ni, int nj, int nk, int nl, DATA_TYPE alpha, DATA_TYPE beta,
             DATA_TYPE POLYBENCH_2D(tmp, NI, NJ, ni, nj),
             DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
             DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
             DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj),
             DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl)) {
  
  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_2mm (ni, nj, nk, nl,
	      alpha, beta,
	      tmp,
	      A,
	      B,
	      C,
	      D);

  /* Stop and print timer. */
  printf("CPU OMP Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;
}

/* ------------------------------------------------------------- */
/* CUDA Version. */
void GPU_argv_init() {
  fprintf(stderr, "GPU init.\n");
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice(GPU_DEVICE);
}

/* ------------------------------------------------------------- */
__global__ void mm2_kernel1(int ni, int nj, int nk, int nl, DATA_TYPE alpha,
                            DATA_TYPE beta, DATA_TYPE *tmp, DATA_TYPE *A,
                            DATA_TYPE *B) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < _PB_NI) && (j < _PB_NJ)) {
    tmp[i * NJ + j] = 0;
    int k;
    for (k = 0; k < _PB_NK; k++) {
      tmp[i * NJ + j] += alpha * A[i * NK + k] * B[k * NJ + j];
    }
  }
}

/* ------------------------------------------------------------- */
__global__ void mm2_kernel2(int ni, int nj, int nk, int nl, DATA_TYPE alpha,
                            DATA_TYPE beta, DATA_TYPE *tmp, DATA_TYPE *C,
                            DATA_TYPE *D) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < _PB_NI) && (j < _PB_NL)) {
    D[i * NL + j] *= beta;
    int k;
    for (k = 0; k < _PB_NJ; k++) {
      D[i * NL + j] += tmp[i * NJ + k] * C[k * NL + j];
    }
  }
}

/* ------------------------------------------------------------- */
void GPU_data_allocation(void){
  fprintf(stderr, "GPU_data_allocation.\n");

  if(!gpu_data_allocated){
    hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * NI * NJ);
    hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
    hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
    hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NL * NJ);
    hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * NI * NL);
    gpu_data_allocated = true;
  }
}

/* ------------------------------------------------------------- */
void GPU_data_copy(DATA_TYPE POLYBENCH_2D(tmp, NI, NJ, ni, nj),
             DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
             DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
             DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj),
             DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl)){

  fprintf(stderr, "GPU_data_copy.\n");

  if(!gpu_data_copied){
    hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
    hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
    hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NL * NJ, hipMemcpyHostToDevice);
    hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);
    gpu_data_copied = true;
  }
}

/* ------------------------------------------------------------- */
void GPU_data_copy_back(DATA_TYPE POLYBENCH_2D(D_outputFromGpu, NI, NL, ni, nl))

hipMemcpy(D_outputFromGpu, D_gpu, sizeof(DATA_TYPE) * NI * NL,
             hipMemcpyDeviceToHost);

}



/* ------------------------------------------------------------- */
/* A caller for each kernel, because OMP generate two for loops structures. 
 * Put the gpu pointer as global, to verify allocations and copies.
*/
void mm2Cuda_1(int ni, int nj, int nk, int nl, DATA_TYPE alpha, DATA_TYPE beta,
             DATA_TYPE POLYBENCH_2D(tmp, NI, NJ, ni, nj),
             DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
             DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
             DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj),
             DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl),
             DATA_TYPE POLYBENCH_2D(D_outputFromGpu, NI, NL, ni, nl)) {
  fprintf(stderr, "mm2Cuda_1.\n");
  
  GPU_argv_init();

  GPU_data_allocation();

  GPU_data_copy();

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid1((size_t)ceil(((float)NJ) / ((float)block.x)),
             (size_t)ceil(((float)NI) / ((float)block.y)));
  
  /* Start timer. */
  polybench_start_instruments;

  mm2_kernel1<<<grid1, block>>>(ni, nj, nk, nl, alpha, beta, tmp_gpu, A_gpu,
                                B_gpu);
  hipDeviceSynchronize();
  
  printf("GPU Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;
}

/* ------------------------------------------------------------- */
void mm2Cuda_2(int ni, int nj, int nk, int nl, DATA_TYPE alpha, DATA_TYPE beta,
             DATA_TYPE POLYBENCH_2D(tmp, NI, NJ, ni, nj),
             DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
             DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
             DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj),
             DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl),
             DATA_TYPE POLYBENCH_2D(D_outputFromGpu, NI, NL, ni, nl)) {
  
  GPU_argv_init();

  GPU_data_allocation();

  GPU_data_copy();

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  
  dim3 grid2((size_t)ceil(((float)NL) / ((float)block.x)),
             (size_t)ceil(((float)NI) / ((float)block.y)));

  /* Start timer. */
  polybench_start_instruments;

  mm2_kernel2<<<grid2, block>>>(ni, nj, nk, nl, alpha, beta, tmp_gpu, C_gpu,
                                D_gpu);
  hipDeviceSynchronize();

  printf("GPU Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;

  GPU_data_copy_back(D_outputFromGpu);
}

/* ------------------------------------------------------------- */
int main(int argc, char **argv) {
  /* Retrieve problem size. */
  int ni = NI;
  int nj = NJ;
  int nk = NK;
  int nl = NL;

  /* Variable declaration/allocation. */
  DATA_TYPE alpha;
  DATA_TYPE beta;
  POLYBENCH_2D_ARRAY_DECL(tmp, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, NI, NK, ni, nk);
  POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, NK, NJ, nk, nj);
  POLYBENCH_2D_ARRAY_DECL(C, DATA_TYPE, NL, NJ, nl, nj);
  POLYBENCH_2D_ARRAY_DECL(D, DATA_TYPE, NI, NL, ni, nl);
  POLYBENCH_2D_ARRAY_DECL(D_outputFromOMP, DATA_TYPE, NI, NL, ni, nl);
  POLYBENCH_2D_ARRAY_DECL(D_outputFromGpu, DATA_TYPE, NI, NL, ni, nl);


  fprintf(stderr, "Preparing alternatives functions.\n");
  /* Preparing the call to target function.
  void mm2Cuda(int ni, int nj, int nk, int nl, DATA_TYPE alpha, DATA_TYPE beta,
             DATA_TYPE POLYBENCH_2D(tmp, NI, NJ, ni, nj),
             DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
             DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
             DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj),
             DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl),
             DATA_TYPE POLYBENCH_2D(D_outputFromGpu, NI, NL, ni, nl))
  */
  // Number of parameters to function.
  int n_params = 12;

  // void handler_function_init_array_GPU(void)
  Func *ff_1 = (Func *) malloc(sizeof(Func));
  Func *ff_2 = (Func *) malloc(sizeof(Func));

  // Number of arguments + 1, the lists need to have last element NULL.
  ff_1->arg_types = (ffi_type**) malloc ((n_params + 1) * sizeof(ffi_type*));
  ff_1->arg_values = (void**) malloc ((n_params + 1) * sizeof(void*));

  ff_2->arg_types = (ffi_type**) malloc ((n_params + 1) * sizeof(ffi_type*));
  ff_2->arg_values = (void**) malloc ((n_params + 1) * sizeof(void*));

  ff_1->f = &mm2Cuda_1;

  memset(&ff_1->ret_value, 0, sizeof(ff_1->ret_value));

  ff_2->f = &mm2Cuda_2;
  memset(&ff_2->ret_value, 0, sizeof(ff_2->ret_value));

  // return type.
  ff_1->ret_type = &ffi_type_void;

  // return type.
  ff_2->ret_type = &ffi_type_void;

  ff_1->nargs = n_params;

  ff_1->arg_values[0] = &ni;
  ff_1->arg_values[1] = &nj;
  ff_1->arg_values[2] = &nk;
  ff_1->arg_values[3] = &nl;
  ff_1->arg_values[4] = &alpha;
  ff_1->arg_values[5] = &beta;
  ff_1->arg_values[6] = &tmp;
  ff_1->arg_values[7] = &A;
  ff_1->arg_values[8] = &B;
  ff_1->arg_values[9] = &C;
  ff_1->arg_values[10] = &D;
  ff_1->arg_values[11] = &D_outputFromGpu;
  ff_1->arg_values[12] = NULL;

  ff_1->arg_types[0] = &ffi_type_sint32;
  ff_1->arg_types[1] = &ffi_type_sint32;
  ff_1->arg_types[2] = &ffi_type_sint32;
  ff_1->arg_types[3] = &ffi_type_sint32;
  ff_1->arg_types[4] = &ffi_type_double;
  ff_1->arg_types[5] = &ffi_type_double;
  ff_1->arg_types[6] = &ffi_type_pointer;
  ff_1->arg_types[7] = &ffi_type_pointer;
  ff_1->arg_types[8] = &ffi_type_pointer;
  ff_1->arg_types[9] = &ffi_type_pointer;
  ff_1->arg_types[10] = &ffi_type_pointer;
  ff_1->arg_types[11] = &ffi_type_pointer;
  ff_1->arg_types[12] = NULL;

  ff_2->nargs = n_params;

  ff_2->arg_values[0] = &ni;
  ff_2->arg_values[1] = &nj;
  ff_2->arg_values[2] = &nk;
  ff_2->arg_values[3] = &nl;
  ff_2->arg_values[4] = &alpha;
  ff_2->arg_values[5] = &beta;
  ff_2->arg_values[6] = &tmp;
  ff_2->arg_values[7] = &A;
  ff_2->arg_values[8] = &B;
  ff_2->arg_values[9] = &C;
  ff_2->arg_values[10] = &D;
  ff_2->arg_values[11] = &D_outputFromGpu;
  ff_2->arg_values[12] = NULL;

  ff_2->arg_types[0] = &ffi_type_sint32;
  ff_2->arg_types[1] = &ffi_type_sint32;
  ff_2->arg_types[2] = &ffi_type_sint32;
  ff_2->arg_types[3] = &ffi_type_sint32;
  ff_2->arg_types[4] = &ffi_type_double;
  ff_2->arg_types[5] = &ffi_type_double;
  ff_2->arg_types[6] = &ffi_type_pointer;
  ff_2->arg_types[7] = &ffi_type_pointer;
  ff_2->arg_types[8] = &ffi_type_pointer;
  ff_2->arg_types[9] = &ffi_type_pointer;
  ff_2->arg_types[10] = &ffi_type_pointer;
  ff_2->arg_types[11] = &ffi_type_pointer;
  ff_2->arg_types[12] = NULL;

  /*          device 0
   * loop 0   mm2Cuda_1
   * loop 1   mm2Cuda_2
   * matrix 2 x 1.
  */
  fprintf(stderr, "Creating table of target functions.\n");
  int nloops = 2;
  int ndevices = 1;

  if (create_target_functions_table(&table, nloops, ndevices)) {
    // Set up the library Functions table.
    assert(table != NULL);

    fprintf(stderr, "Declaring function in 0,0.\n");
    table[0][0][0] = *ff_1;
    table[1][0][0] = *ff_2;

    TablePointerFunctions = table;
    assert(TablePointerFunctions != NULL);
  }

  fprintf(stderr, "Calling init_array.\n");
  /* Initialize array(s). */
  init_array(ni, nj, nk, nl, &alpha, &beta, POLYBENCH_ARRAY(A),
             POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D));
  
  fprintf(stderr, "Calling gemm_original.\n");
  mm_original(ni, nj, nk, nl, alpha, beta, POLYBENCH_ARRAY(tmp), POLYBENCH_ARRAY(A),
          POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D));

  fprintf(stderr, "Calling gemm_omp.\n");
  mm_omp(ni, nj, nk, nl,
	      alpha, beta,
	      POLYBENCH_ARRAY(tmp),
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(B),
	      POLYBENCH_ARRAY(C),
	      POLYBENCH_ARRAY(D_outputFromOMP));

  fprintf(stderr, "Calling compareResults(original, omp).\n");
  compareResults(ni, nl, POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(D_outputFromOMP));

  fprintf(stderr, "Calling gemm_cuda.\n");  
  mm2Cuda_1(ni, nj, nk, nl, alpha, beta, POLYBENCH_ARRAY(tmp), POLYBENCH_ARRAY(A),
          POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D),
          POLYBENCH_ARRAY(D_outputFromGpu));

  // fprintf(stderr, "Calling gemm_cuda using Table of Pointers.\n");
  // call_function_ffi_call(table[0][0]);

  fprintf(stderr, "Calling compareResults(original, cuda).\n");
  compareResults(ni, nl, POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(D_outputFromGpu));


  polybench_prevent_dce(print_array(ni, nl, POLYBENCH_ARRAY(D_outputFromGpu)));

  POLYBENCH_FREE_ARRAY(tmp);
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);
  POLYBENCH_FREE_ARRAY(C);
  POLYBENCH_FREE_ARRAY(D);
  POLYBENCH_FREE_ARRAY(D_outputFromOMP);
  POLYBENCH_FREE_ARRAY(D_outputFromGpu);


  if(gpu_data_allocated){
  	hipFree(tmp_gpu);
  	hipFree(A_gpu);
  	hipFree(B_gpu);
  	hipFree(C_gpu);
  	hipFree(D_gpu);	
  }

  return 0;
}

#include <polybench.c>