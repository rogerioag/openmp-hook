#include "hip/hip_runtime.h"
/**
 * syr2k.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

#define POLYBENCH_TIME 1

#include "syr2k.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

// Offloading support functions.
#include <offload.h>

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

#define RUN_ON_CPU

/* ------------------------------------------------------------- */
void init_arrays(int ni, int nj, DATA_TYPE *alpha, DATA_TYPE *beta,
                 DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
                 DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
                 DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni)) {
  int i, j;

  *alpha = 32412;
  *beta = 2123;

  for (i = 0; i < ni; i++) {
    for (j = 0; j < nj; j++) {
      A[i][j] = ((DATA_TYPE)i * j) / ni;
      B[i][j] = ((DATA_TYPE)i * j) / ni;
    }
  }

  for (i = 0; i < ni; i++) {
    for (j = 0; j < ni; j++) {
      C[i][j] = ((DATA_TYPE)i * j) / ni;
    }
  }
}

/* ------------------------------------------------------------- */
void copy_array(int ni, DATA_TYPE POLYBENCH_2D(C_source, NI, NI, ni, ni), DATA_TYPE POLYBENCH_2D(C_dest, NI, NI, ni, ni)) {
  int i, j;

  for (i = 0; i < ni; i++) {
    for (j = 0; j < ni; j++) {
      C_dest[i][j] = C_source[i][j];
      // printf("%4.2f - %4.2f\n", C_dest[i][j], C_source[i][j]);
    }
  }
}


/* ------------------------------------------------------------- */
void syr2kCpu(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
              DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni)) {
  int i, j, k;

  /*    C := alpha*A*B' + alpha*B*A' + beta*C */
  for (i = 0; i < _PB_NI; i++) {
    for (j = 0; j < _PB_NI; j++) {
      C[i][j] *= beta;
    }
  }

  for (i = 0; i < _PB_NI; i++) {
    for (j = 0; j < _PB_NI; j++) {
      for (k = 0; k < _PB_NJ; k++) {
        C[i][j] += alpha * A[i][k] * B[j][k];
        C[i][j] += alpha * B[i][k] * A[j][k];
      }
    }
  }
}

/* ------------------------------------------------------------- */
void syr2k_original(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
              DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
              DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni)) {
  
  /* Start timer. */
  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  syr2kCpu(ni, nj, alpha, beta, A, B, C);

  /* Stop and print timer. */
  printf("Original CPU Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;
}

/* ------------------------------------------------------------- */
/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static void syr2k_omp_kernel(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
                         DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
                         DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
                         DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni)) {
  int i, j, k;

  #pragma scop
  current_loop_index = 0;
  #pragma omp parallel
  {
    /*    C := alpha*A*B' + alpha*B*A' + beta*C */
    #pragma omp for private(j) schedule(runtime)
    for (i = 0; i < _PB_NI; i++)
      for (j = 0; j < _PB_NI; j++)
        C[i][j] *= beta;
    #pragma omp for private(j, k) schedule(runtime)
    for (i = 0; i < _PB_NI; i++)
      for (j = 0; j < _PB_NI; j++)
        for (k = 0; k < _PB_NJ; k++) {
          C[i][j] += alpha * A[i][k] * B[j][k];
          C[i][j] += alpha * B[i][k] * A[j][k];
        }
  }
  #pragma endscop
}

/* ------------------------------------------------------------- */
void syr2k_omp(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
                         DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
                         DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
                         DATA_TYPE POLYBENCH_2D(C_outputFromOMP, NI, NI, ni, ni)) {

  /* Start timer. */
  polybench_start_instruments;

  syr2k_omp_kernel(ni, nj, alpha, beta, A, B, C_outputFromOMP);

  /* Stop and print timer. */
  printf("OpenMP Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;
}

/* ------------------------------------------------------------- */
void compareResults(int ni, DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni),
                    DATA_TYPE POLYBENCH_2D(C_output, NI, NI, ni, ni)) {
  int i, j, fail;
  fail = 0;

  // Compare C with D
  for (i = 0; i < ni; i++) {
    for (j = 0; j < ni; j++) {
      // printf("%4.2f - %4.2f\n", C[i][j], C_output[i][j]);
      if (percentDiff(C[i][j], C_output[i][j]) >
          PERCENT_DIFF_ERROR_THRESHOLD) {
        fail++;
      }
    }
  }

  // print results
  printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f "
         "Percent: %d\n",
         PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

/* ------------------------------------------------------------- */
void GPU_argv_init() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice(GPU_DEVICE);
}

/* ------------------------------------------------------------- */
__global__ void syr2k_cuda_kernel(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
                             DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NI) && (j < NI)) {
    c[i * NI + j] *= beta;

    int k;
    for (k = 0; k < NJ; k++) {
      c[i * NI + j] += alpha * a[i * NJ + k] * b[j * NJ + k] +
                       alpha * b[i * NJ + k] * a[j * NJ + k];
    }
  }
}

/* ------------------------------------------------------------- */
void syr2k_cuda(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
               DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
               DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
               DATA_TYPE POLYBENCH_2D(C_inputToGpu, NI, NI, ni, ni),
               DATA_TYPE POLYBENCH_2D(C_outputFromGpu, NI, NI, ni, ni)) {

  GPU_argv_init();

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;
  DATA_TYPE *C_gpu;

  hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NI);
  hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
  hipMemcpy(C_gpu, C_inputToGpu, sizeof(DATA_TYPE) * NI * NI, hipMemcpyHostToDevice);

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((size_t)ceil(((float)NI) / ((float)DIM_THREAD_BLOCK_X)),
            (size_t)(ceil(((float)NI) / ((float)DIM_THREAD_BLOCK_Y))));

  /* Start timer. */
  polybench_start_instruments;

  syr2k_cuda_kernel<<<grid, block>>>(ni, nj, alpha, beta, A_gpu, B_gpu, C_gpu);
  hipDeviceSynchronize();

  /* Stop and print timer. */
  printf("GPU Time in seconds:\n");
  polybench_stop_instruments;
  polybench_print_instruments;

  hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NI,
             hipMemcpyDeviceToHost);

  hipFree(A_gpu);
  hipFree(B_gpu);
  hipFree(C_gpu);
}

/* ------------------------------------------------------------- */
/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni)) {
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < ni; j++) {
      fprintf(stderr, DATA_PRINTF_MODIFIER, C[i][j]);
      if ((i * ni + j) % 20 == 0)
        fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}

/* ------------------------------------------------------------- */
int main(int argc, char *argv[]) {
  /* Retrieve problem size. */
  int ni = NI;
  int nj = NJ;

  /* Variable declaration/allocation. */
  DATA_TYPE alpha;
  DATA_TYPE beta;
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(C, DATA_TYPE, NI, NI, ni, ni);
  POLYBENCH_2D_ARRAY_DECL(C_outputFromOMP, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(C_inputToGpu, DATA_TYPE, NI, NJ, ni, nj);
  POLYBENCH_2D_ARRAY_DECL(C_outputFromGpu, DATA_TYPE, NI, NJ, ni, nj);

  fprintf(stderr, "Preparing alternatives functions.\n");
  /* Preparing the call to target function.
  void syr2k_cuda(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta,
               DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj),
               DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
               DATA_TYPE POLYBENCH_2D(C_inputToGpu, NI, NI, ni, ni),
               DATA_TYPE POLYBENCH_2D(C_outputFromGpu, NI, NI, ni, ni))
  */
  // Number of parameters to function.
  int n_params = 8;

  // void handler_function_init_array_GPU(void)
  Func *ff_1 = (Func *) malloc(sizeof(Func));

  // Number of arguments + 1, the lists need to have last element NULL.
  ff_1->arg_types = (ffi_type**) malloc ((n_params + 1) * sizeof(ffi_type*));
  ff_1->arg_values = (void**) malloc ((n_params + 1) * sizeof(void*));

  ff_1->f = &syr2k_cuda;
  memset(&ff_1->ret_value, 0, sizeof(ff_1->ret_value));

  // return type.
  ff_1->ret_type = &ffi_type_void;

  ff_1->nargs = n_params;

  ff_1->arg_values[0] = &ni;
  ff_1->arg_values[1] = &nj;
  ff_1->arg_values[2] = &alpha;
  ff_1->arg_values[3] = &beta;
  ff_1->arg_values[4] = &A;
  ff_1->arg_values[5] = &B;
  ff_1->arg_values[6] = &C_inputToGpu;
  ff_1->arg_values[7] = &C_outputFromGpu;
  ff_1->arg_values[8] = NULL;

  ff_1->arg_types[0] = &ffi_type_sint32;
  ff_1->arg_types[1] = &ffi_type_sint32;
  ff_1->arg_types[2] = &ffi_type_double;
  ff_1->arg_types[3] = &ffi_type_double;
  ff_1->arg_types[4] = &ffi_type_pointer;
  ff_1->arg_types[5] = &ffi_type_pointer;
  ff_1->arg_types[6] = &ffi_type_pointer;
  ff_1->arg_types[7] = &ffi_type_pointer;
  ff_1->arg_types[8] = NULL;

  /*          device 0
   * loop 0   gemm_cuda
   * matrix 1 x 1.
  */
  fprintf(stderr, "Creating table of target functions.\n");
  int nloops = 1;
  int ndevices = 1;

  if (create_target_functions_table(&table, nloops, ndevices)) {
    // Set up the library Functions table.
    assert(table != NULL);

    fprintf(stderr, "Declaring function in 0,0.\n");
    table[0][0][0] = *ff_1;

    TablePointerFunctions = table;
    assert(TablePointerFunctions != NULL);
  }

  fprintf(stderr, "Calling init_array.\n");
  init_arrays(ni, nj, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B),
              POLYBENCH_ARRAY(C));

  /*Copy the original C to C of OMP.*/
  fprintf(stderr, "Copying C to C_outputFromOMP.\n");
  // memcpy(C_outputFromOMP, C, sizeof(C_outputFromOMP));
  copy_array(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromOMP));
  
  // printf("%4.2f - %4.2f\n", *(C[0][0]), *(C_outputFromOMP[0][0]));
  // compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromOMP));

  fprintf(stderr, "Copying C to C_outputFromOMP.\n");
  // memcpy(C_inputToGpu, C, sizeof(C_inputToGpu));
  copy_array(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));
  // compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

  fprintf(stderr, "Calling Original.\n");
  syr2k_original(ni, nj, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));

  fprintf(stderr, "Calling OMP.\n");
  syr2k_omp(ni, nj, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C_outputFromOMP));

  fprintf(stderr, "Calling compareResults(original, omp).\n");
  compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromOMP));

  // fprintf(stderr, "Calling CUDA.\n");
  // syr2k_cuda(ni, nj, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C_inputToGpu), POLYBENCH_ARRAY(C_outputFromGpu));

  // fprintf(stderr, "Calling gemm_cuda using Table of Pointers.\n");
  // call_function_ffi_call(table[0][0]);

  fprintf(stderr, "Calling compareResults(original, cuda).\n");
  compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));
  
  polybench_prevent_dce(print_array(ni, POLYBENCH_ARRAY(C_outputFromGpu)));

  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);
  POLYBENCH_FREE_ARRAY(C);
  POLYBENCH_FREE_ARRAY(C_outputFromOMP);
  POLYBENCH_FREE_ARRAY(C_outputFromGpu);

  return 0;
}

#include <polybench.c>