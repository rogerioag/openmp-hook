#include "hip/hip_runtime.h"
/**
 * jacobi2D.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <stdlib.h>
#include <stdarg.h>
#include <math.h>

#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "jacobi2D.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

// Macros to generate openmp schedule.
#include <macros.h>

// Offloading support functions.
#include <offload.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define RUN_ON_CPU

/* ------------------------------------------------------------- */
/* Arrays initialization. */
void init_array(int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(B,N,N,n,n))
{
	int i, j;

	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
		{
			A[i][j] = ((DATA_TYPE) i*(j+2) + 10) / N;
			B[i][j] = ((DATA_TYPE) (i-4)*(j-1) + 11) / N;
		}
	}
}

/* ------------------------------------------------------------- */
void compareResults(int n, DATA_TYPE POLYBENCH_2D(a,N,N,n,n), DATA_TYPE POLYBENCH_2D(a_outputFromGpu,N,N,n,n), DATA_TYPE POLYBENCH_2D(b,N,N,n,n), DATA_TYPE POLYBENCH_2D(b_outputFromGpu,N,N,n,n))
{
	int i, j, fail;
	fail = 0;   

	// Compare output from CPU and GPU
	for (i=0; i<n; i++) 
	{
		for (j=0; j<n; j++) 
		{
			if (percentDiff(a[i][j], a_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
        }
	}
  
	for (i=0; i<n; i++) 
	{
       	for (j=0; j<n; j++) 
		{
        		if (percentDiff(b[i][j], b_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
        			fail++;
        		}
       	}
	}

	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

/* ------------------------------------------------------------- */
/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_2D(A,N,N,n,n))

{
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++) {
      fprintf(stderr, DATA_PRINTF_MODIFIER, A[i][j]);
      if ((i * n + j) % 20 == 0) fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}

/* ------------------------------------------------------------- */
void runJacobi2DCpu(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(B,N,N,n,n)) {
	
	for (int t = 0; t < _PB_TSTEPS; t++) {
    	for (int i = 1; i < _PB_N - 1; i++) {
			for (int j = 1; j < _PB_N - 1; j++) {
	  			B[i][j] = 0.2f * (A[i][j] + A[i][(j-1)] + A[i][(1+j)] + A[(1+i)][j] + A[(i-1)][j]);
			}
		}
		
    	for (int i = 1; i < _PB_N-1; i++) {
			for (int j = 1; j < _PB_N-1; j++) {
	  			A[i][j] = B[i][j];
			}
		}
	}
}

/* ------------------------------------------------------------- */
void jacobi2d_original(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(B,N,N,n,n)){

	/* Start timer. */
	polybench_start_instruments;

	runJacobi2DCpu(tsteps, n, A, B);
	
	/* Stop and print timer. */
	polybench_stop_instruments;
	printf("CPU Time in seconds:\n");
	polybench_print_instruments;
}

/* ------------------------------------------------------------- */
/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void jacobi_2d_imper_omp_kernel(int tsteps,
			    int n,
			    DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
			    DATA_TYPE POLYBENCH_2D(B,N,N,n,n))
{
  int t, i, j;

  #pragma scop
  
  #pragma omp parallel private(i,j,t) num_threads(OPENMP_NUM_THREADS)
  {
    #pragma omp master
    {
      for (t = 0; t < _PB_TSTEPS; t++) {
        #pragma omp for schedule(OPENMP_SCHEDULE_WITH_CHUNK) 
        for (i = 1; i < _PB_N - 1; i++)
          for (j = 1; j < _PB_N - 1; j++)
            B[i][j] = 0.2 * (A[i][j] + A[i][j-1] + A[i][1+j] + A[1+i][j] + A[i-1][j]);
	    
	      #pragma omp for schedule(OPENMP_SCHEDULE_WITH_CHUNK) 
        for (i = 1; i < _PB_N-1; i++)
          for (j = 1; j < _PB_N-1; j++)
            A[i][j] = B[i][j];
      }
    }
  }

  #pragma endscop
}

/* ------------------------------------------------------------- */
void jacobi_2d_imper_omp(int tsteps,
			    int n,
			    DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
			    DATA_TYPE POLYBENCH_2D(B,N,N,n,n)){
	/* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  jacobi_2d_imper_omp_kernel (tsteps, n, A, B);

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

}

/* ------------------------------------------------------------- */
__global__ void jacobi_cuda_kernel_1(int n, DATA_TYPE* A, DATA_TYPE* B)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if ((i >= 1) && (i < (_PB_N-1)) && (j >= 1) && (j < (_PB_N-1)))
	{
		B[i*N + j] = 0.2f * (A[i*N + j] + A[i*N + (j-1)] + A[i*N + (1 + j)] + A[(1 + i)*N + j] + A[(i-1)*N + j]);	
	}
}

/* ------------------------------------------------------------- */
__global__ void jacobi_cuda_kernel_2(int n, DATA_TYPE* A, DATA_TYPE* B)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((i >= 1) && (i < (_PB_N-1)) && (j >= 1) && (j < (_PB_N-1)))
	{
		A[i*N + j] = B[i*N + j];
	}
}

/* ------------------------------------------------------------- */
void jacobi2d_cuda(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(B,N,N,n,n), DATA_TYPE POLYBENCH_2D(A_outputFromGpu,N,N,n,n), DATA_TYPE POLYBENCH_2D(B_outputFromGpu,N,N,n,n))
{
	DATA_TYPE* Agpu;
	DATA_TYPE* Bgpu;

	hipMalloc(&Agpu, N * N * sizeof(DATA_TYPE));
	hipMalloc(&Bgpu, N * N * sizeof(DATA_TYPE));
	hipMemcpy(Agpu, A, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
	hipMemcpy(Bgpu, B, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((unsigned int)ceil( ((float)N) / ((float)block.x) ), (unsigned int)ceil( ((float)N) / ((float)block.y) ));
	
	/* Start timer. */
  	polybench_start_instruments;

	for (int t = 0; t < _PB_TSTEPS; t++)
	{
		jacobi_cuda_kernel_1<<<grid,block>>>(n, Agpu, Bgpu);
		hipDeviceSynchronize();

		jacobi_cuda_kernel_2<<<grid,block>>>(n, Agpu, Bgpu);
		hipDeviceSynchronize();
	}

	/* Stop and print timer. */
  	polybench_stop_instruments;
  	printf("GPU Time in seconds:\n");
  	polybench_print_instruments;
	
	hipMemcpy(A_outputFromGpu, Agpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);
	hipMemcpy(B_outputFromGpu, Bgpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);

	hipFree(Agpu);
	hipFree(Bgpu);
}

/* ------------------------------------------------------------- */
void copy_array(int n, DATA_TYPE POLYBENCH_2D(source, N, N, n, n), DATA_TYPE POLYBENCH_2D(dest, N, N, n, n)) {
  int i, j;

  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      dest[i][j] = source[i][j];
    }
  }
}

/* ------------------------------------------------------------- */
int main(int argc, char** argv)
{
	/* Retrieve problem size. */
	int n = N;
	int tsteps = TSTEPS;

	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,N,N,n,n);
	POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,N,N,n,n);
	POLYBENCH_2D_ARRAY_DECL(A_outputFromGpu,DATA_TYPE,N,N,n,n);
	POLYBENCH_2D_ARRAY_DECL(B_outputFromGpu,DATA_TYPE,N,N,n,n);

	POLYBENCH_2D_ARRAY_DECL(A_inputToGpu,DATA_TYPE,N,N,n,n);
	POLYBENCH_2D_ARRAY_DECL(B_inputToGpu,DATA_TYPE,N,N,n,n);

	init_array(n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));

	copy_array(n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(A_inputToGpu));
	copy_array(n, POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_inputToGpu));
	
	// void jacobi2d_cuda(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(B,N,N,n,n), 
	//                    DATA_TYPE POLYBENCH_2D(A_outputFromGpu,N,N,n,n), DATA_TYPE POLYBENCH_2D(B_outputFromGpu,N,N,n,n))

	// Number of parameters to function.
  int n_params = 6;

  // loop 0.
  Func *ff_0 = (Func *) malloc(sizeof(Func));

  // Number of arguments + 1, the lists need to have last element NULL.
  ff_0->arg_types = (ffi_type**) malloc ((n_params + 1) * sizeof(ffi_type*));
  ff_0->arg_values = (void**) malloc ((n_params + 1) * sizeof(void*));

  ff_0->f = &jacobi2d_cuda;
  memset(&ff_0->ret_value, 0, sizeof(ff_0->ret_value));

  // return type.
  ff_0->ret_type = &ffi_type_void;

  ff_0->nargs = n_params;

  ff_0->arg_values[0] = &tsteps;
  ff_0->arg_values[1] = &n;
  ff_0->arg_values[2] = &A;
  ff_0->arg_values[3] = &B;
  ff_0->arg_values[4] = &A_outputFromGpu;
  ff_0->arg_values[5] = &B_outputFromGpu;
  ff_0->arg_values[6] = NULL;

  ff_0->arg_types[0] = &ffi_type_sint32;
  ff_0->arg_types[1] = &ffi_type_sint32;
  ff_0->arg_types[2] = &ffi_type_pointer;
  ff_0->arg_types[3] = &ffi_type_pointer;
  ff_0->arg_types[4] = &ffi_type_pointer;
  ff_0->arg_types[5] = &ffi_type_pointer;
  ff_0->arg_types[6] = NULL;

  // loop 1.
  Func *ff_1 = (Func *) malloc(sizeof(Func));

  // Number of arguments + 1, the lists need to have last element NULL.
  ff_1->arg_types = (ffi_type**) malloc ((n_params + 1) * sizeof(ffi_type*));
  ff_1->arg_values = (void**) malloc ((n_params + 1) * sizeof(void*));

  ff_1->f = &syr2k_cuda_1;
  memset(&ff_1->ret_value, 0, sizeof(ff_1->ret_value));

  // return type.
  ff_1->ret_type = &ffi_type_void;

  ff_1->nargs = n_params;

  ff_1->arg_values[0] = &ni;
  ff_1->arg_values[1] = &nj;
  ff_1->arg_values[2] = &alpha;
  ff_1->arg_values[3] = &beta;
  ff_1->arg_values[4] = &A;
  ff_1->arg_values[5] = &B;
  ff_1->arg_values[6] = &C_inputToGpu;
  ff_1->arg_values[7] = &C_outputFromGpu;
  ff_1->arg_values[8] = NULL;

  ff_1->arg_types[0] = &ffi_type_sint32;
  ff_1->arg_types[1] = &ffi_type_sint32;
  ff_1->arg_types[2] = &ffi_type_double;
  ff_1->arg_types[3] = &ffi_type_double;
  ff_1->arg_types[4] = &ffi_type_pointer;
  ff_1->arg_types[5] = &ffi_type_pointer;
  ff_1->arg_types[6] = &ffi_type_pointer;
  ff_1->arg_types[7] = &ffi_type_pointer;
  ff_1->arg_types[8] = NULL;

  /*          device 0
   * loop 0   gemm_cuda
   * matrix 1 x 1.
  */
  fprintf(stderr, "Creating table of target functions.\n");
  int nloops = 2;
  int ndevices = 2;

  if (create_target_functions_table(&table, nloops, ndevices)) {
    // Set up the library Functions table.
    assert(table != NULL);

    fprintf(stderr, "Declaring function in 0,0.\n");
    table[0][1][0] = *ff_0;
    table[1][1][0] = *ff_1;

    TablePointerFunctions = table;
    assert(TablePointerFunctions != NULL);
  }

  fprintf(stderr, "Calling init_array.\n");
  init_arrays(ni, nj, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B),
              POLYBENCH_ARRAY(C));

  /*Copy the original C to C of OMP.*/
  fprintf(stderr, "Copying C to C_outputFromOMP.\n");
  // memcpy(C_outputFromOMP, C, sizeof(C_outputFromOMP));
  copy_array(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromOMP));
  
  // printf("%4.2f - %4.2f\n", *(C[0][0]), *(C_outputFromOMP[0][0]));
  // compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromOMP));

  fprintf(stderr, "Copying C to C_outputFromOMP.\n");
  // memcpy(C_inputToGpu, C, sizeof(C_inputToGpu));
  copy_array(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));
  // compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

  fprintf(stderr, "Calling Original.\n");
  syr2k_original(ni, nj, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));

  fprintf(stderr, "Calling OMP.\n");
  syr2k_omp(ni, nj, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C_outputFromOMP));

  fprintf(stderr, "Calling compareResults(original, omp).\n");
  compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromOMP));

  // fprintf(stderr, "Calling CUDA.\n");
  // syr2k_cuda(ni, nj, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C_inputToGpu), POLYBENCH_ARRAY(C_outputFromGpu));

  // fprintf(stderr, "Calling gemm_cuda using Table of Pointers.\n");
  // call_function_ffi_call(table[0][0]);

  fprintf(stderr, "Calling compareResults(original, cuda).\n");
  compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

  polybench_prevent_dce(print_array(ni, POLYBENCH_ARRAY(C_outputFromGpu)));






	compareResults(n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(a_outputFromGpu), POLYBENCH_ARRAY(b), POLYBENCH_ARRAY(b_outputFromGpu));

	polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(a_outputFromGpu)));

	
	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(A_outputFromGpu);
	POLYBENCH_FREE_ARRAY(B);
	POLYBENCH_FREE_ARRAY(B_outputFromGpu);

	return 0;
}

// polybench.c uses the OpenMP to parallelize somethings. This call were intercepted by hookomp.
#undef _OPENMP

#include <polybench.c>